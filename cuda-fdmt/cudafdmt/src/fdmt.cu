#include "hip/hip_runtime.h"

#include "cpu_kernels.h"
#include "gpu_kernels.h"
#include "cuda_fdmt.h"
#include "fdmt.h"


float dm_delay(const float f1, const float f2) {
  return 4.14e9*(isquaref(f1) - isquaref(f2));
}

__host__ __device__ float squaref(const float f)
{
  return f*f;
}

__host__ __device__ float isquaref(const float f)
{
  return 1.0f/(f*f);
}



__host__ __device__ float cff(float f1_start, float f1_end, float f2_start, float f2_end)
{
  float rf = (isquaref(f1_start) - isquaref(f1_end))/(isquaref(f2_start) - isquaref(f2_end));
  //  printf("rff %f %f %f %f %f\n", f1_start, f1_end, f2_start, f2_end, rf);

  return rf;
}

__host__ __device__ int calc_delta_t(const fdmt_t* fdmt, float f_start, float f_end)
{
  float rf = cff(f_start, f_end, fdmt->fmin, fdmt->fmax);
  float delta_tf = ((float)fdmt->max_dt-1.0) * rf;
  int delta_t = (int)ceilf(delta_tf);

  //  printf("delta t: rf %f delta_tf %f delta_t %d\n", rf, delta_tf, delta_t);
  return delta_t;
}

int fdmt_create(fdmt_t* fdmt, float fmin, float fmax, int nf, int max_dt, int nbeams)
{
  fdmt->max_dt = max_dt;
  fdmt->fmin = fmin;
  fdmt->fmax = fmax;
  fdmt->nf = nf;
  fdmt->df = (fdmt->fmax - fdmt->fmin)/((float) fdmt->nf);
  fdmt->order = (int)ceil(log(fdmt->nf)/log(2.0));
  fdmt->nbeams = nbeams;
  assert(nf > 0);
  assert(max_dt > 0);
  assert(1<<fdmt->order >= fdmt->nf);
  assert(nbeams >= 1);

  // TODO: CHeck it's important that fmin < fmax??
  assert(fmin > 0);
  assert(fmax > 0);

  //deltaT = int(np.ceil((maxDT-1) *(1./f_min**2 - 1./(f_min + deltaF)**2) / (1./f_min**2 - 1./f_max**2)))

  //fdmt->delta_t = (int)(ceilf((fdmt->maxDT-1) *(isquaref(fdmt->f_min) - isquaref(fdmt->f_min + fdmt->delta_f)) / (isquaref(f_min) - isquaref(f_max))));

  // Delta_t here is the number of time samples the maximum DM trajectory traverses
  // In the lowest channel. It is equivalent to the number of Dm trials you need to do
  // In the lowest channel to get out to the highest DM we asked for.
  fdmt->delta_t = calc_delta_t(fdmt, fdmt->fmin, fdmt->fmin + fdmt->df);
  fdmt->delta_t += 1; // Siglhtly different definition to origiinal

  // Allocate states as ping-pong buffer
  fdmt->state_size = fdmt->nbeams * fdmt->nf*fdmt->delta_t * fdmt->max_dt;
  fdmt->state_nbytes = fdmt->state_size * sizeof(fdmt_dtype);
  for (int s = 0; s < 2; s++) {
    fdmt->states[s].nw = fdmt->nbeams;
    fdmt->states[s].nx = fdmt->max_dt;
    fdmt->states[s].ny = fdmt->delta_t;
    fdmt->states[s].nz = fdmt->nf;
    array4d_malloc(&fdmt->states[s]);
  }

  return 0;
}


int fdmt_initialise(const fdmt_t* fdmt, array3d_t* indata, array4d_t* state)
{

  // indata is 3D array: (nbeams, nf, nt)
  // State is a 4D array: (nbeams, nf, deltat, nt) ( for the moment)

  assert(indata->nx == fdmt->nbeams);
  assert(indata->ny == fdmt->nf);
  assert(indata->nz == fdmt->max_dt);

  state->nw = fdmt->nbeams;
  state->nx = fdmt->nf;
  state->ny = fdmt->delta_t;
  state->nz = fdmt->max_dt;

  // zero off the state
  bzero(state->d, state->nw*state->nx*state->ny*state->nz*sizeof(fdmt_dtype));

  // Assign initial data to the state at delta_t=0
  for(int beam = 0 ; beam < fdmt->nbeams; beam++) {
	  for (int c = 0; c < fdmt->nf; c++) {
		int outidx = array4d_idx(state, beam, c, 0, 0);
		int inidx = array3d_idx(indata, beam, c, 0);
		for (int t = 0; t < fdmt->max_dt; t++) {
		  state->d[outidx + t] = indata->d[inidx + t];
		}
	  }
  }

  // do partial sums initialisation (Equation 20.)
  // This (like everything barak does) is done as a recursive sum

  for(int beam = 0; beam < fdmt->nbeams; beam++) {
	  // For each frequency channel
	  for (int c = 0; c < fdmt->nf; c++) {
		// For each delta_t, i.e. each single-channel DM trial
		for (int idt = 1; idt < fdmt->delta_t; idt++) {
		  int outidx = array4d_idx(state, beam, c, idt, 0);
		  int iidx = array4d_idx(state, beam, c, idt-1, 0);
		  int imidx = array3d_idx(indata, beam, c, indata->ny - 1);

		  // The state for dt=d = the state for dt=(d-1) + the time-reversed input sample
		  // for each time
		  // (TODO: Not including a missing overlap here)
		  for (int j = idt; j < fdmt->max_dt; j++) {
			state->d[outidx + j] = state->d[iidx + j] + indata->d[imidx - j];
		  }
		}
	  }
  }

  return 0;

}


int fdmt_iteration(const fdmt_t* fdmt,
                   const int iteration_num,
                   const array4d_t* indata,
                   array4d_t* outdata)
{
  float df = fdmt->df; // channel resolution
  float delta_f = (float)(1 << iteration_num) * df; // Resolution of current iteration
  int delta_t = calc_delta_t(fdmt, fdmt->fmin, fdmt->fmin+delta_f); // Max DM

  // Outdata has size (nbeams, o_nf, o_nd1, fdmt->nt)
  outdata->nw = indata->nw;
  outdata->nx = indata->nx/2 + indata->nx % 2; // Add 1 to the frequency dimension if it's not divisible by 2
  outdata->ny = delta_t + 1;
  outdata->nz = indata->nz;

  assert(array4d_size(outdata) <= fdmt->state_size);

  //    printf("iteration %d df %f delta_f %f delta_t %d output nx=%d ny=%d nz%d\n",
  //           iteration_num, df, delta_f, delta_t, outdata->nx, outdata->ny, outdata->nz);

  // zero that output baby
  bzero(outdata->d, outdata->nw*outdata->nx * outdata->ny * outdata->nz * sizeof(fdmt_dtype));
  array4d_cuda_memset(outdata, 0);

  int shift_input = 0; // ?
  int shift_output = 0; // ?

  float fjumps = (float)outdata->nx; // Output number of channels
  float frange = fdmt->fmax - fdmt->fmin; // Width of band
  float fmin = fdmt->fmin; // Bottom of band

  float correction = 0.0;
  if (iteration_num > 0) {
    correction = df/2.0;
  }

  assert(indata->nw == fdmt->nbeams);
  // For each output sub-band
  for (int iif = 0; iif < outdata->nx; iif++) {
	float f_start = frange/fjumps * (float)iif + fmin; // Top freq of subband
	float f_end = frange/fjumps*((float)iif + 1) + fmin; // Bottom freq of subband
	float f_middle = (f_end - f_start)/2.0 + f_start - correction; // Middle freq of subband, less 0.5xresolution
	float f_middle_larger = (f_end - f_start)/2.0 + f_start + correction; // Middle freq of subband + 0.5x resolution (helps with rounding)

	// Max DM for this subband
	int delta_t_local = calc_delta_t(fdmt, f_start, f_end) + 1;

	// For each DM relevant for this subband
	for (int idt = 0; idt < delta_t_local; idt++) {
	  int dt_middle = roundf(idt * cff(f_middle, f_start, f_end, f_start)); // Dt for middle freq less 0.5xresolution
	  int dt_middle_index = dt_middle + shift_input;
	  int dt_middle_larger = roundf(idt * cff(f_middle_larger, f_start, f_end, f_start)); // Dt for middle freq +0.5x resolution
	  int dt_rest = idt - dt_middle_larger;
	  int dt_rest_index = dt_rest + shift_input;

	  int itmin = 0;
	  int itmax = dt_middle_larger;

	  //Output[i_F,i_dT + ShiftOutput,i_T_min:i_T_max] = Input[2*i_F, dT_middle_index,i_T_min:i_T_max];
	  //int outidx = array4d_idx(outdata, beam, iif, idt+shift_output, 0);
	  //int inidx1  = array4d_idx(indata, beam, 2*iif, dt_middle_index, 0);


	  //printf("iteration %d channel %d freq %f idt %d dt_local "
		//	  "%d dt_middle %d dt_middle_larger %d dt_rest %d\n",
		//	  iteration_num, iif, f_middle, idt, delta_t_local, dt_middle_index, dt_middle_larger, dt_rest_index);

	  // Here we handle the edge effects and set
	  // OUtput state[freq, idx, 0:dtmin] = input_state[2xfreq, dt_middle, 0:dtmin]
	  // where the DM would have overun the available times
	  // This needs to be fixed for more careful time overlapping
	  coord3_t dst_start = {.x = iif, .y = idt+shift_output, .z = 0};
	  coord3_t src1_start = {.x = 2*iif, .y = dt_middle_index, .z = 0};
	  array_gpu_copy1(outdata, indata, &dst_start, &src1_start, dt_middle_larger);
	  //cpu_copy2(&outdata->d[outidx + itmin], &indata->d[inidx1 + itmin], (itmax - itmin));
	  //for (int i = itmin; i < itmax; i++) {
	  //outdata->d[outidx + i] = indata->d[inidx1 + i];
	  //}


	  // Now we work on the remaining times that are guaranteed not to overrun the input dimensions
	  itmin = dt_middle_larger;
	  itmax = fdmt->max_dt;

	  coord3_t src2_start = {.x = 2*iif + 1, .y = dt_rest_index, .z = 0};
	  // src and dst now start from a bit offset
	  src1_start.z = dt_middle_larger;
	  dst_start.z = dt_middle_larger;
      int zcount = itmax - itmin;


	  if (2*iif + 1 < indata->nx) { // If the input data has this channel, we'll add it in
		//Output[i_F,i_dT + ShiftOutput,i_T_min:i_T_max] = Input[2*i_F, dT_middle_index,i_T_min:i_T_max] + Input[2*i_F+1, dT_rest_index,i_T_min - dT_middle_larger:i_T_max-dT_middle_larger]
		// playinga trick here - we're always addign the fastest moving index
		// Putting -dt_middle_larger in array3d_idx would have caused an assertion failure
		// But ofsetting by dt_middle_larger at the end, we get the best of all worlds

		//int inidx2 = array4d_idx(indata, beam, 2*iif+1, dt_rest_index, 0) - dt_middle_larger;

		array_gpu_sum1(outdata, indata, &dst_start, &src1_start, &src2_start, zcount);

		//for(int i = itmin; i < itmax; i++) {
		//  outdata->d[outidx + i] = indata->d[inidx1 + i] + indata->d[inidx2 + i];
		//}
		//cpu_sum1(&outdata->d[outidx + itmin], &indata->d[inidx1+itmin], &indata->d[inidx2+itmin], itmax-itmin);


	  } else { // Just copy the input over. which basically assumes the upper channel is flaggedd/0
		  // TODO: Could probably be done outside the iif loop to save evalutating IFs, but
		  // Too tricky for the moment.
		//cpu_copy2(&outdata->d[outidx + itmin], &indata->d[inidx1 + itmin], (itmax - itmin));
		//for(int i = itmin; i < itmax; i++) {
		//  outdata->d[outidx + i] = indata->d[inidx1 + i];
		//	}
		  array_gpu_copy1(outdata, indata, &dst_start, &src1_start, zcount);
	  }
	}

  }
  return 0;
}

int fdmt_execute(fdmt_t* fdmt, fdmt_dtype* indata, fdmt_dtype* outdata)
{
  array3d_t inarr = {.nx = fdmt->nbeams, .ny = fdmt->nf, .nz = fdmt->max_dt};
  inarr.d = indata;

  // Make the final outstate - this saves a memcpy on the final iteration
  array4d_t outstate;
  outstate.nw = fdmt->states[0].nw;
  outstate.nx = fdmt->states[0].nx;
  outstate.ny = fdmt->states[0].ny;
  outstate.nz = fdmt->states[0].nz;
  outstate.d = outdata;

  // Start that puppy up
  int s = 0;
  fdmt_initialise(fdmt, &inarr, &fdmt->states[s]);
  array4d_copy_to_device(&fdmt->states[s]);

#ifdef DUMP_STATE
  char buf[128];
  sprintf(buf, "state_s%d.dat", 0);
  array4d_dump(&fdmt->states[s], buf);
#endif

  for (int iter = 1; iter < fdmt->order+1; iter++) {
    //printf("Iteration %d\n", iter);
    array4d_t* currstate = &fdmt->states[s];
    array4d_t* newstate;

    // If it's the last iteration, cheekily substitute the output pointer
    // to save a memcopy
    // if (iter == fdmt->order) {
    s = (s + 1) % 2;
    if (iter == fdmt->order) {
      newstate = &outstate;
      newstate->d_device = fdmt->states[s].d_device;
      printf("Setting outstate\n");
    } else {
      newstate = &fdmt->states[s];
    }
    fdmt_iteration(fdmt, iter, currstate, newstate);
    cuda_fdmt_iteration<<<fdmt->nbeams, fdmt->max_dt >>>(*fdmt, iter, *currstate, *newstate);
#ifdef DUMP_STATE
    array4d_copy_to_host(newstate);
    sprintf(buf, "state_s%d.dat", iter);
    array4d_dump(newstate, buf);
#endif



    //printf("Finisehd iteration %d\n", iter);

  }

  //printf("Returing form execute\n");

  return 0;
}
