/*
 * fftbench.cu
 *
 *  Created on: 17 Sep 2018
 *      Author: ban115

 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "cuda_utils.h"
#include "cufft_utils.h"

__device__ hipfftComplex load_callback(void *dataIn,
				      size_t offset,
				      void *callerInfo,
				      void *sharedPtr) {
  hipfftComplex value = {0.0f, 0.0f};
  return value;
}
__device__ hipfftCallbackLoadC d_loadCallbackPtr = load_callback;

__device__ void store_callback(void *dataOut,
			       size_t offset,
			       hipfftReal element,
			       void *callerInfo,
			       void *sharedPtr) {
}
__device__ hipfftCallbackStoreR d_storeCallbackPtr = store_callback;


template <class intype>
void timefft(int n, int batch, hipDataType itype, hipDataType etype, hipDataType otype, bool inplace)
{
  CudaTimer t;
  intype *data, *out_data;
  hipfftHandle plan;
  size_t data_size=sizeof(intype)*n*(n/2 + 1)*batch;
  //size_t data_size=sizeof(intype)*n*n*batch;
  gpuErrchk(hipMalloc((void**) &data, data_size));
  if (inplace) {
    out_data = data;
  } else {
    gpuErrchk(hipMalloc((void**) &out_data, data_size));
  }

  long long int nsize[] = {n,n };

  size_t worksize;
  cufftSafeCall(hipfftCreate(&plan));
  cufftSafeCall(hipfftXtMakePlanMany(plan, 2, nsize,
  				    NULL, 1, 0, itype,
  				    NULL, 1, 0, otype,
  				    batch, &worksize, etype
  				    ));

  /*
   * Retrieve address of callback functions on the device
   */
  hipfftCallbackLoadR h_loadCallbackPtr;
  hipfftCallbackStoreR h_storeCallbackPtr;
  gpuErrchk(hipMemcpyFromSymbol(&h_loadCallbackPtr,
  				 HIP_SYMBOL(d_loadCallbackPtr), 
  				 sizeof(h_loadCallbackPtr)));
  gpuErrchk(hipMemcpyFromSymbol(&h_storeCallbackPtr,
				 HIP_SYMBOL(d_storeCallbackPtr),
				 sizeof(h_storeCallbackPtr)));
  
  // Now associate the callbacks with the plan.
  hipfftResult status = hipfftXtSetCallback(plan, 
  					  (void **)&h_loadCallbackPtr, 
  					  HIPFFT_CB_LD_COMPLEX,
  					  0);
  if (status == CUFFT_LICENSE_ERROR) {
    fprintf(stdout, "This sample requires a valid license file.\n");
    fprintf(stdout, "The file was either not found, out of date, or otherwise invalid.\n");
    exit(EXIT_FAILURE);
  } else {
    cufftSafeCall(status);
  }
  cufftSafeCall(hipfftXtSetCallback(plan,
  				   (void **)&h_storeCallbackPtr,
  				   HIPFFT_CB_ST_REAL,
  				   NULL));
  
  // warm up
  cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));

  int niter = 100;
  for (int i = 0; i < niter; ++i) {
    t.start();
    //cufftSafeCall(hipfftExecC2R(plan, data, (outtype*) data));
    cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));
    t.stop();
  }
  
  float tavg_us = t.get_average_time() / float(batch) * 1e3f;

  printf("%dx%d FFT batch=%d data=%d MB in-place=%d type=%d-> %d. Worksize=%d MB: %f microseconds/FFT= %f k FFTs/sec total=%0.2fs\n",
	 n,n,batch,data_size/1024/1024, inplace, itype,otype, worksize/1024/1024, tavg_us, 1./tavg_us*1e6f/1e3f);
  cufftSafeCall(hipfftDestroy(plan));
  gpuErrchk(hipFree(data));
  if (! inplace) {
    gpuErrchk(hipFree(out_data));
  }
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("%s Usage: gpuid N batchmin batchmax\n", argv[0]);
    return EXIT_FAILURE;
  }

  int cuda_device = atoi(argv[1]);
  hipDeviceProp_t p;
  gpuErrchk(hipGetDeviceProperties(&p, cuda_device));

  printf("FFT Benchmark \n");
  printf("Device[%d]=%s v%d.%d Mem=%d GB shmem/block=%d constmem=%d Warp=%d Clock=%d MHz %d multiprocessors\n",
	 cuda_device, p.name, p.major, p.minor, p.totalGlobalMem/1024/1024/1024, p.sharedMemPerBlock,
	 p.totalConstMem, p.warpSize, p.clockRate/1000, p.multiProcessorCount
	 );

  gpuErrchk( hipSetDevice(cuda_device));

  int n = atoi(argv[2]);
  int batchmin = atoi(argv[3]);
  int batchmax = atoi(argv[4]);
  hipDataType itype = HIP_C_32F;
  hipDataType etype = HIP_C_32F;
  hipDataType otype = HIP_R_32F;

  for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
    int batch = 1 << batch2;
    timefft<hipfftComplex>(n,batch,itype, etype, otype, false);
  }

  for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
    int batch = 1 << batch2;
    timefft<hipfftComplex>(n,batch,itype, etype, otype, true);
  }
  
  //itype = HIP_C_16F;
  //etype = HIP_C_16F;
  //otype = HIP_R_16F;
  //
  //for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
  //  int batch = 1 << batch2;
  //  timefft<half2>(n,batch,itype, etype, otype, false);
  //}
  //
  //for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
  //  int batch = 1 << batch2;
  //  timefft<half2>(n,batch,itype, etype, otype, true);
  //}

  printf("Benchmark finished\n");
}
