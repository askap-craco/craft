/*
 * fftbench.cu
 *
 *  Created on: 17 Sep 2018
 *      Author: ban115

 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "cuda_utils.h"
#include "cufft_utils.h"


//typedef half2 intype;
//typedef half outtype;

//typedef hipfftComplex intype;
//typedef hipfftReal outtype;
//typedef hipfftComplex ftype;

template <class intype>
void timefft(int n, int batch, hipDataType itype, hipDataType etype, hipDataType otype, bool inplace)
{
	CudaTimer t;
	intype *data, *out_data;
	hipfftHandle plan;
	size_t data_size=sizeof(intype)*n*(n/2 + 1)*batch;
	gpuErrchk(hipMalloc((void**) &data, data_size));
	if (inplace) {
	  out_data = data;
	} else {
	  gpuErrchk(hipMalloc((void**) &out_data, data_size));
	}

	long long int nsize[] = {n,n };

		/*cufftSafeCall(hipfftPlanMany(&plan, 2, n,
			NULL, 1, 0, // Simple input layout
			NULL, 1, 0, // Simple output layout
			HIPFFT_C2R, BATCH));
	*/
	size_t worksize;
	cufftSafeCall(hipfftCreate(&plan));
	//int gpus[]  = { cuda_device };
	//cufftSafeCall(hipfftXtSetGPUs(plan, 1, gpus));
	//cufftSafeCall(hipfftSetAutoAllocation());

	//hipDataType itype = HIP_C_16F;
	//hipDataType etype = HIP_C_16F;
	//hipDataType otype = HIP_R_16F;
	cufftSafeCall(hipfftXtMakePlanMany(plan, 2, nsize,
			NULL, 1, 0, itype,
			NULL, 1, 0, otype,
			batch, &worksize, etype
			));

	// warm up
	cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));

	int niter = 100;
	for (int i = 0; i < niter; ++i) {
		t.start();
		//cufftSafeCall(hipfftExecC2R(plan, data, (outtype*) data));
		cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));
		t.stop();

	}
	float tavg_us = t.get_average_time() / float(batch) * 1e3f;

	printf("%dx%d FFT batch=%d data=%d MB in-place=%d type=%d-> %d. Worksize=%d MB: %f microseconds/FFT= %f k FFTs/sec total=%0.2fs\n",
	       n,n,batch,data_size/1024/1024, inplace, itype,otype, worksize/1024/1024, tavg_us, 1./tavg_us*1e6f/1e3f);
	cufftSafeCall(hipfftDestroy(plan));
	gpuErrchk(hipFree(data));
	if (! inplace) {
	  gpuErrchk(hipFree(out_data));
	}
}

int main(int argc, char* argv[])
{

	if (argc != 5) {
		printf("%s Usage: gpuid N batchmin batchmax\n", argv[0]);
		return EXIT_FAILURE;
	}

	int cuda_device = atoi(argv[1]);
	hipDeviceProp_t p;
	gpuErrchk(hipGetDeviceProperties(&p, cuda_device));

	printf("FFT Benchmark \n");
	printf("Device[%d]=%s v%d.%d Mem=%d GB shmem/block=%d constmem=%d Warp=%d Clock=%d MHz %d multiprocessors\n",
			cuda_device, p.name, p.major, p.minor, p.totalGlobalMem/1024/1024/1024, p.sharedMemPerBlock,
			p.totalConstMem, p.warpSize, p.clockRate/1000, p.multiProcessorCount
			);

	gpuErrchk( hipSetDevice(cuda_device));

	int n = atoi(argv[2]);
	int batchmin = atoi(argv[3]);
	int batchmax = atoi(argv[4]);
	hipDataType itype = HIP_C_32F;
	hipDataType etype = HIP_C_32F;
	hipDataType otype = HIP_R_32F;

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<hipfftComplex>(n,batch,itype, etype, otype, false);
	}

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<hipfftComplex>(n,batch,itype, etype, otype, true);
	}


	itype = HIP_C_16F;
	etype = HIP_C_16F;
	otype = HIP_R_16F;

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<half2>(n,batch,itype, etype, otype, false);
	}

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<half2>(n,batch,itype, etype, otype, true);
	}


	printf("Benchmark finished\n");
}



