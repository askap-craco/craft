/*
 * CudaTimer.cpp
 *
 *  Created on: 4 Oct 2016
 *      Author: ban115
 */

#include "CudaTimer.h"
#include "cuda_utils.h"

CudaTimer::CudaTimer(hipStream_t stream) {
	gpuErrchk(hipEventCreate(&m_start));
	gpuErrchk(hipEventCreate(&m_stop));
	m_stream = stream;
}

CudaTimer::~CudaTimer() {
	gpuErrchk(hipEventDestroy(m_start));
	gpuErrchk(hipEventDestroy(m_stop));
}

void CudaTimer::start() {
	gpuErrchk(hipEventRecord(m_start, m_stream));
}

void CudaTimer::stop() {
	gpuErrchk(hipEventRecord(m_stop, m_stream));
	sync_stop();

}

void CudaTimer::sync_start() {
	gpuErrchk(hipEventSynchronize(m_start));
}

void CudaTimer::sync_stop() {
	gpuErrchk(hipEventSynchronize(m_stop));
}

float CudaTimer::get_elapsed_time() {
	float ms;
	gpuErrchk(hipEventElapsedTime(&ms, m_start, m_stop));
	return ms;
}
