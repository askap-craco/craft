/*
 * CudaTimer.cpp
 *
 *  Created on: 4 Oct 2016
 *      Author: ban115
 */

#include "CudaTimer.h"
#include "cuda_utils.h"

CudaTimer::CudaTimer(hipStream_t stream) {
	gpuErrchk(hipEventCreate(&m_start));
	gpuErrchk(hipEventCreate(&m_stop));

	m_stream = stream;
	m_total_time = 0;
	m_ncalls = 0;
}

CudaTimer::~CudaTimer() {
	gpuErrchk(hipEventDestroy(m_start));
	gpuErrchk(hipEventDestroy(m_stop));
}

void CudaTimer::start() {
	gpuErrchk(hipEventRecord(m_start, m_stream));
	cputimer.start();
}

void CudaTimer::stop() {
	gpuErrchk(hipEventRecord(m_stop, m_stream));
	sync_stop();
	m_total_time += get_elapsed_time();
	m_ncalls += 1;
	cputimer.stop();
}

void CudaTimer::sync_start() {
	gpuErrchk(hipEventSynchronize(m_start));
}

void CudaTimer::sync_stop() {
	gpuErrchk(hipEventSynchronize(m_stop));
}

float CudaTimer::get_elapsed_time() {
	// Returns elapsed time in milliseconds.
	float ms;
	if (m_ncalls == 0) { // if it hasn't been called, hipEventElapsedTime fails with invalidResourceHandle
		ms = 0;
	} else {
		gpuErrchk(hipEventElapsedTime(&ms, m_start, m_stop));
	}
	return ms;
}

float CudaTimer::get_average_time() {
	// average elapsed time per clal in ms
	return m_total_time/(float)m_ncalls;
}
