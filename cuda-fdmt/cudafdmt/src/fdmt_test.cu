#include "hip/hip_runtime.h"
//
//  fdmt_test.c
//  fdmt
//
//  Created by Keith Bannister on 19/07/2016.
//  Copyright (c) 2016 Keith Bannister. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include "fdmt.h"
#include "cuda_fdmt.h"
#include "CudaTimer.h"


using namespace std;

void usage() {
	fprintf(stderr,
			"fdmt_test [options] infile outfile\n"
			"	-d Number of dispersion trials\n"
			"	-t Samples per block\n"
			"	-f Number of frequency channels\n"
			"	-b Number of beams\n"
			"	-x Maximum frequency (MHz)\n"
			"	-h Print this message\n"
	);
	exit(EXIT_FAILURE);
}

int main(int argc, char* argv[])
{
	int nd = 512;
	int nt = 256;
	int nf = 512;
	int nbeams = 1;
	float fmax = 1440;
	char ch;
	while ((ch = getopt(argc, argv, "d:t:f:b:x:g:h")) != -1) {
		switch (ch) {
		case 'd':
			nd = atoi(optarg);
			break;
		case 't':
			nt = atoi(optarg);
			break;
		case 'f':
			nf = atoi(optarg);
			break;
		case 'b':
			nbeams = atoi(optarg);
			break;
		case 'x':
			fmax = atof(optarg);
			break;
		case '?':
		case 'h':
		default:
			usage();
		}
	}
	argc -= optind;
	argv += optind;

	float fmin = fmax - (float)nf;

	int blockin = nf*nt;
	int blockout = nd*nt;
	fdmt_dtype* din = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockin*nbeams);
	fdmt_dtype* din_tmp = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockin*nbeams);
	fdmt_dtype* dout = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockout*nbeams);
	printf("Starting! fmin=%f fmax=%f nbeams=%d nf=%d nd=%d nt=%d\n", fmin, fmax, nbeams, nf, nd, nt);

	if (argc != 2) {
		printf("Not enough arguments\n");
		exit(EXIT_FAILURE);
	}

	FILE* fin = fopen(argv[0], "r");
	if (fin == NULL) {
		perror("Could not open input file");
		exit(EXIT_FAILURE);
	}

	FILE* fout = fopen(argv[1], "w");
	if (fout == NULL) {
		perror("Could not open output file");
		exit(EXIT_FAILURE);
	}


	fdmt_t fdmt;
	fdmt_create(&fdmt, fmin, fmax, nf, nd, nt, nbeams);

	// read input file until exhausted
	while (fread(din_tmp, sizeof(fdmt_dtype), blockin, fin) == blockin) {

		// File is in TF format. We need FT order.
		// Do the transpose
		for(int t = 0; t < nt; ++t) {
			for (int f = 0; f < nf; f++) {
				din[f*nt + t] = din_tmp[f + nf*t];
			}
		}
		// copy to all beams
		for(int b = 1; b < nbeams; b++) {
			int idx = b*blockin;
			//memcpy(&din[idx], din, blockin*sizeof(fdmt_dtype));
		}

		CudaTimer t;
		t.start();
		for(int i = 0; i < 1; i++) {
			fdmt_execute(&fdmt, din, dout);
		}
		t.stop();
		cout << "FDMT Execute loop took " << t << endl;
		fwrite(dout, sizeof(fdmt_dtype), blockout, fout);
		cout << "Wrote " << blockout << " elements to outfile. First two are:" << dout[0] << dout[1] << endl;
	}
	fclose(fin);
	fclose(fout);
}
