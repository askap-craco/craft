#include "hip/hip_runtime.h"
//
//  fdmt_test.c
//  fdmt
//
//  Created by Keith Bannister on 19/07/2016.
//  Copyright (c) 2016 Keith Bannister. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <signal.h>
#include <limits.h>
#include <float.h>
#include <omp.h>
#include <sys/time.h>
#include <sys/resource.h>
#include "fdmt.h"
#include "array.h"
#include "boxcar.h"
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "DataSource.h"
#include "SigprocFile.h"
#include "SigprocFileSet.h"
#include "DataSource.h"
#include "DadaSource.h"
#include "DadaSet.h"
#include "FilDirSet.h"
#include "CandidateList.h"
#include "InvalidSourceFormat.h"
#include "Rescaler.h"
#include "rescale.h"
#include "DadaSink.h"


using namespace std;

void runtest_usage() {
	fprintf(stderr,
			"cudafdmt [options] [infile [infile[ ...]]\n"
			"   -d D - Number of dispersion trials. Negative D computes negative DMs\n"
			"   -t T - Samples per block\n"
			"   -s S - Decay timescale\n"
			"   -o FILE - Candidate filename\n"
			"   -U host:port - UDP host:port to send candidates to\n"
			"   -x SN - threshold S/N\n"
			"   -D dump intermediate data to disk (SLOW)\n"
			"   -R dump rescaler data to disk\n"
			"   -B b - Process b beams simultaneously to save memory\n"
			"   -r R - Blocks per rescale update (0 for no rescaling)\n"
			"   -S S - Seek to this number of seconds before starting\n"
			"   -M M - Channel Mean relative change threshold (0.2 is OK)\n"
			"   -T T - Channel StdDev relative changed flagging threshold (0.2 is OK)\n"
			"   -K K - Channel Kurtosis threshold (3 is pretty good)\n"
//			"   -G N - Channel flag channel growing (flags N channels either side of a bad channel)\n"
			"   -z Z - Zap times with 0 DM above threshold Z\n"
			"   -C C - Zap time/frequency cells with S/N above threshold C\n"
			"   -u   - Subtract DM0 time series from spectrum\n"
			"   -p   - Sum polarisations\n"
			"   -n ncand - Maximum mumber of candidates to write per block\n"
			"   -m mindm - Minimum DM to report candidates for (to ignore 0 DM junk)\n"
			"   -b maxbc - Maximum boxcar to create a candidate. Candidates with peaks above this boxcar are ignored\n"
			"   -g G - CUDA device\n"
			"   -N N - Maximum number of blocks to process before quitting\n"
			"   -X x - Export incoherent sum data to this DADA key\n"
			"   -h Print this message\n"
			"    Version: %s\n"
			, VERSION);
	exit(EXIT_FAILURE);
}

volatile bool stopped;

//typedef void (*sig_t) (int);

void handle_signal(int signal)
{
	stopped = true;
}

void dumparr(const char* prefix, const int blocknum, array4d_t* arr, bool copy=true)
{
	char fbuf[1024];
	sprintf(fbuf, "%s_e%d.dat", prefix, blocknum);
	if (copy) {
		array4d_copy_to_host(arr);
	}
	int nz = 0;
	int size = array4d_size(arr);
	for(int i = 0; i < size; i++) {
		if (arr->d[i] == 0.0) {
			nz += 1;
		}
	}

	//printf("Dumping %s %s %d zeros\n", prefix, fbuf, nz);
	array4d_dump(arr, fbuf);
}

void dump_rescaler(int iblock, Rescaler* rescaler)
{
	dumparr("mean", iblock, &rescaler->mean);
	dumparr("std", iblock, &rescaler->std);
	dumparr("kurt", iblock, &rescaler->kurt);
	dumparr("nsamps", iblock, &rescaler->nsamps);
	dumparr("dm0", iblock, &rescaler->dm0);
	dumparr("dm0count", iblock, &rescaler->dm0count);
	dumparr("dm0stats", iblock, &rescaler->dm0stats);
	dumparr("scale", iblock, &rescaler->scale);
	dumparr("offset", iblock, &rescaler->offset);
	dumparr("decay_offset", iblock, &rescaler->decay_offset);
}

int main(int argc, char* argv[])
{
	int nd = 1024;
	int nt = 512;
	float seek_seconds = 0.0;
	int num_rescale_blocks = 2;
	float decay_timescale = 1.0; // Seconds?
	char ch;
	float thresh = 10.0;
	const char* out_filename = "fredda.cand";
	bool dump_data = false;
	bool do_dump_rescaler = false;
	int cuda_device = 0;
	float kurt_thresh = INFINITY;
	float std_thresh = INFINITY;
	float mean_thresh = INFINITY;
	float dm0_thresh = INFINITY;
	float cell_thresh = INFINITY;
	int flag_grow = 3;
	int max_ncand_per_block = 4096;
	int mindm = 0;
	int maxbc = 32;
	int max_nblocks = INT_MAX;
	int nbeams_alloc = -1;
	bool subtract_dm0 = false;
	bool polsum = false;
	char udp_host[128];
	bzero(udp_host, 128);
	short udp_port = -1;
	int export_dada_key = -1;

	printf("Fredda version %s starting. Cmdline: ", VERSION);
	for (int c = 0; c < argc; ++c) {
		printf("%s ", argv[c]);
	}
	printf("\n");

	while ((ch = getopt(argc, argv, "d:t:s:o:x:r:S:B:DRg:M:T:U:K:G:C:n:m:b:z:N:X:uhp")) != -1) {
		switch (ch) {
		case 'd':
			nd = atoi(optarg);
			break;
		case 't':
			nt = atoi(optarg);
			break;
		case 's':
			decay_timescale = atof(optarg);
			break;
		case 'o':
			out_filename = optarg;
			break;
		case 'x':
			thresh = atof(optarg);
			break;
		case 'D':
			dump_data = true;
			break;
		case 'R':
			do_dump_rescaler = true;
			break;
		case 'r':
			num_rescale_blocks = atoi(optarg);
			break;
		case 'S':
			seek_seconds = atof(optarg);
			break;
		case 'g':
			cuda_device = atoi(optarg);
			break;
		case 'K':
			kurt_thresh = atof(optarg);
			break;
		case 'T':
			std_thresh = atof(optarg);
			break;
		case 'M':
			mean_thresh = atof(optarg);
			break;
		case 'G':
			flag_grow = atoi(optarg);
			break;
		case 'C':
			cell_thresh = atof(optarg);
			break;
		case 'n':
			max_ncand_per_block = atoi(optarg);
			break;
		case 'm':
			mindm = atoi(optarg);
			break;
		case 'u':
			subtract_dm0 = true;
			break;
		case 'b':
			maxbc = atoi(optarg);
			break;
		case 'z':
			dm0_thresh = atof(optarg);
			break;
		case 'N':
			max_nblocks = atoi(optarg);
			break;
		case 'p':
			polsum = true;
			break;
		case 'B':
			nbeams_alloc = atoi(optarg);
			break;
		case 'X':
			sscanf(optarg, "%x",&export_dada_key);
			break;
		case 'U':
		{
			char* colon = strchr(optarg, ':');
			if (colon == NULL) {
				printf("Invalid hostport\n");
				exit(EXIT_FAILURE);
			}
			memcpy(udp_host, optarg, colon-optarg);
			udp_port = atoi(colon+1);
		}
		break;

		case '?':
		case 'h':
		default:
			runtest_usage();
		}
	}
	argc -= optind;
	argv += optind;

	if (argc == 0) {
		printf("Not enough arguments: %d\n");
		exit(EXIT_FAILURE);
	}

	printf("\n");
	printf("Setting cuda device to %d\n", cuda_device);
	gpuErrchk( hipSetDevice(cuda_device));

	CpuTimer tall;
	CpuTimer tproc;
	CudaTimer trescale;
	CudaTimer tboxcar;
	tall.start();

	DataSource* source = NULL;
	DadaSet* dada_source = NULL; // for debugging
	try {
		// load sigproc file
		SigprocFileSet* fs_source = new SigprocFileSet(nt, argc, argv);
		source = fs_source;
	} catch (InvalidSourceFormat& e) {
		try {
			dada_source = new DadaSet(nt, argc, argv);
			source = dada_source;
		} catch (InvalidSourceFormat& e) {
			try {
				source = new FilDirSet(nt, argc, argv);
			} catch (InvalidSourceFormat& e) {
				printf("No valid inputs\n");
				exit(EXIT_FAILURE);
			}
		}
	}
	assert(seek_seconds >= 0);
	int num_skip_blocks = seek_seconds / source->tsamp() / nt;
	printf("Seeking to start of data: block %d nsamples=%d time=%fs\n", num_skip_blocks, num_skip_blocks*nt, num_skip_blocks*nt*source->tsamp());
	if (num_skip_blocks > 0) {
		source->seek_sample(nt*num_skip_blocks);
	}

	assert(source != NULL);
	bool negdm = (nd < 0);
	CandidateSink sink(source, out_filename, negdm, udp_host, udp_port);
	cout << "spf tsamp " << source->tsamp()<< " ants " << source->nants() << " nbeams " << source->nbeams()
			<< " npols "<< source->npols() << " fch1 " << source->fch1() << " nchans "
			<< source->nchans() << " foff " << source->foff() << endl;
	int nbeams_per_antenna = source->nbeams()*source->npols(); // number of beams including polarisations
	int nbeams_in_total = nbeams_per_antenna*source->nants();
	int npols_in = source->npols();
	int nbeams_out, npols_out;
	if (polsum) { // assume polsum and antsum
		nbeams_out = source->nbeams();
		npols_out = 1;
		assert(nbeams_per_antenna %2 == 0);
	} else { // ant sum only
		nbeams_out = source->nbeams()*source->npols();
		npols_out = source->npols();
	}
	float nbeams_summed = (float(nbeams_in_total)/float(nbeams_out));
	int nf = source->nchans();
	int nbits = source->nbits();
	printf("S/N Threshold %f Max ncand per block %d mindm %d \n", thresh, max_ncand_per_block, mindm);
	//rescale input buffer
	size_t in_buffer_bytes_per_ant = nbeams_per_antenna*nf*nt*nbits/8;
	uint8_t* in_buffer_device;
	printf("Copy in buffer size = %d MB per ant = %d MB TOTAL \n", in_buffer_bytes_per_ant/(1024l*1024l), in_buffer_bytes_per_ant*source->nants()/(1024l*1024l));
	gpuErrchk( hipMalloc((void**) &in_buffer_device, in_buffer_bytes_per_ant*source->nants() ));

	float foff =  (float) source->foff();
	float fmax = (float) source->fch1() - foff; // The FDMT seems to want this offset to make sense of the world. Not sure why.
	float fmin = fmax + nf*foff;


	if (nd < 0) { // Flip the band to calculate negative DMs
		nd = -nd; // make nd positive -otherwise array sizes get confuddled
		// FDMT requres fmin < fmax
		// rescaling will invert the channels now that we've changed the sign of foff
		foff = -foff;
	}

	DadaSink* dada_sink = NULL;
	if (export_dada_key != -1) {
		char* hdr = NULL;
		if (dada_source != NULL) {
			hdr = dada_source->get_source_at(0)->get_header();
		}
		dada_sink = new DadaSink(*source, export_dada_key, hdr, npols_out, nbeams_out, nt);
	}

	// rescale output buffer
	array4d_t rescale_buf;
	rescale_buf.nw = nbeams_out;
	rescale_buf.nx = nf;
	rescale_buf.ny = 1;
	rescale_buf.nz = nt;
	array4d_malloc(&rescale_buf, dump_data, true);

	// rescale junk buffer for first integration only - bleah
	array4d_t rescale_junk_buf;
	rescale_junk_buf.nw = nbeams_out;
	rescale_junk_buf.nx = nf;
	rescale_junk_buf.ny = 1;
	rescale_junk_buf.nz = nt;
	array4d_malloc(&rescale_junk_buf, false, true);

	// FDMT output buffer
	array4d_t out_buf;
	out_buf.nw = nbeams_out;
	out_buf.nx = 1;
	out_buf.ny = nd;
	out_buf.nz = nt;
	array4d_malloc(&out_buf, dump_data, true);


	// create rescaler
	RescaleOptions rescale = {};
	rescale.interval_samps = nt;
	rescale.target_mean = 0.0;
	rescale.target_stdev = 1.0/sqrt(nbeams_summed);
	rescale.decay_constant = 0.35 * decay_timescale / source->tsamp(); // This is how the_decimator.C does it, I think.
	rescale.mean_thresh = mean_thresh;
	rescale.std_thresh = std_thresh;
	rescale.kurt_thresh = kurt_thresh;
	rescale.flag_grow = flag_grow;
	rescale.dm0_thresh = dm0_thresh;
	rescale.cell_thresh = cell_thresh;
	rescale.invert_freq = (foff < 0);
	rescale.subtract_dm0 = subtract_dm0;
	rescale.nt = nt;
	rescale.nf = nf;
	rescale.nbeams_per_ant = nbeams_per_antenna;
	rescale.nants = source->nants();
	rescale.polsum = polsum;
	rescale.nbits = source->nbits();
	rescale.in_order = source->data_order();
	// set guess of initial scale and offset to dm0 thresholding works
	printf("Rescaling to mean=%f stdev=%f decay constant=%f mean/std/kurtosis/dm0/Cell thresholds: %0.1f/%0.1f/%0.1f/%0.1f/%0.1f grow flags by %d channels\n",
			rescale.target_mean,rescale.target_stdev,
			rescale.decay_constant,
			rescale.mean_thresh, rescale.std_thresh, rescale.kurt_thresh,
			rescale.dm0_thresh, rescale.cell_thresh,
			rescale.flag_grow);
	Rescaler* rescaler = new Rescaler(rescale);
	rescaler->set_scaleoffset(1.0f, 0.0f); // Just pass it straight through without rescaling

	float flag_freqs_mhz[] = {1111.0f, 1144.0f};
	int num_flag_freqs = sizeof(flag_freqs_mhz) / sizeof(float);
	for (int flagi = 0; flagi < num_flag_freqs; flagi++) {
		//float freq = source->fch1() + c * source->foff();
		float freq = flag_freqs_mhz[flagi];
		int channel = int(roundf((freq - source->fch1())/source->foff()));
		if (channel >= 0 && channel < nf) {
			printf("Flagging channel %d at frequency %f\n", channel, freq);
			rescaler->flag_channel(channel);
		}
	}

	// Create fdmt
	fdmt_t fdmt;
	printf("Creating FDMT fmin=%f fmax=%f nf=%d nd=%d nt=%d nbeams=%d nbeams_alloc=%d\n",
			fmin, fmax, nf, nd, nt, nbeams_out, nbeams_alloc);
	fdmt_create(&fdmt, fmin, fmax, nf, nd, nt, nbeams_out, nbeams_alloc, dump_data);

	int blocknum = 0;
	int iblock = num_skip_blocks;
	unsigned long long total_candidates = 0;
	unsigned long long num_candidate_overflow_blocks = 0;
	// make boxcar history
	array4d_t boxcar_history;
	boxcar_history.nw = 1;
	boxcar_history.nx = nbeams_out;
	boxcar_history.ny = nd;
	boxcar_history.nz = NBOX;
	array4d_malloc(&boxcar_history, dump_data, true);
	array4d_zero(&boxcar_history);
	// make boxcar discards
	array4d_t boxcar_discards;
	boxcar_discards.nw = 1;
	boxcar_discards.nx = 1;
	boxcar_discards.ny = nbeams_out;
	boxcar_discards.nz = nd;
	array4d_malloc(&boxcar_discards, true, true);
	array4d_cuda_memset(&boxcar_discards, 0);

	// make boxcar output.
	// TODO: Only allocate on GPU if we'll be dumping it to dis.
	// Otherwise, we'll just use candidate lists and save on a bucketload of memory
	array4d_t boxcar_data;
	boxcar_data.nw = nbeams_out;
	boxcar_data.nx = nd;
	boxcar_data.ny = nt;
	boxcar_data.nz = NBOX;
	array4d_malloc(&boxcar_data, dump_data, dump_data);
	array4d_zero(&boxcar_data);

	CandidateList candidate_list(max_ncand_per_block);

	// measure bytes used
	size_t gpu_free_bytes, gpu_total_bytes;
	gpuErrchk(hipMemGetInfo( &gpu_free_bytes, &gpu_total_bytes ));

	// add signal handler
	signal(SIGHUP, &handle_signal);
	signal(SIGINT, &handle_signal);
	signal(SIGTERM, &handle_signal);
	uint64_t num_flagged_beam_chans = 0;
	uint64_t num_flagged_times = 0;

	// Create streams - one for each antenan
	const int MAX_NANT = 72;
	hipStream_t streams[MAX_NANT];
	assert(source->nants() <= MAX_NANT);
	for (int i = 0; i < source->nants(); i++) {
		gpuErrchk(hipStreamCreate(&streams[i]));
		//streams[i] = 0;
	}

	while (true) {
		if (stopped) {
			printf("Stopped due to signal received\n");
			break;
		}
		if (blocknum >= max_nblocks) {
			break;
		}

		rescaler->reset(rescale_buf); // set output buffer to zero - each rescale update will add the result into the buffer

		fdmt.t_copy_in.start();

//#pragma omp parallel
		for(int iant = 0; iant < source->nants(); iant++) {
			// read samples from input - one antenna at a time.
			void* read_buf;
			int this_nt = source->read_samples_ant(&read_buf, iant);
			if (this_nt != nt) { // WE've run out of samples
				stopped = true;
				break;
			}
			// File is in TBF order
			// Output needs to be BFT order
			// Do transpose and cast to float on the way through using GPU
			uint8_t* this_ant_buffer = in_buffer_device + iant*in_buffer_bytes_per_ant;
			gpuErrchk(hipMemcpyAsync(this_ant_buffer,
					read_buf, in_buffer_bytes_per_ant*sizeof(uint8_t), hipMemcpyHostToDevice, streams[iant]));
			//tproc.start();
			//trescale.start();
			if (blocknum == 0 && num_rescale_blocks > 0) { // if first block rescale and update with no
				// flagging so we can work out roughly what the scales are
				// Send output to junk buffer - silly but will fix later
				// TODO: Remove junk buffer to save memory
				rescaler->update_and_transpose(rescale_junk_buf, this_ant_buffer, rescaler->noflag_options, iant, streams[iant]);

				// update scale and offset
				rescaler->update_scaleoffset(rescaler->noflag_options, iant, streams[iant]);
				if (do_dump_rescaler) {
					dump_rescaler(-1, rescaler);
				}
			}

			// this time we rescale with the flagging turned on
			rescaler->update_and_transpose(rescale_buf, this_ant_buffer, rescaler->options, iant, streams[iant]);
			//trescale.stop();
			//tproc.stop();
		}
		gpuErrchk(hipDeviceSynchronize()); // Synchonize after doing all those asynchronous, multistream things
		fdmt.t_copy_in.stop();

		if (stopped) {// if we've run out of samples
			break;
		}

		if (dump_data) {
			dumparr("inbuf", iblock, &rescale_buf);
		}
		// Do asynchronous copy to dada output using the copy stream for antenna 0
		if (dada_sink != NULL) {
			void* outptr = dada_sink->open_block();
			gpuErrchk(hipMemcpyAsync(outptr,
					rescale_buf.d_device,
					array4d_size(&rescale_buf)*sizeof(rescale_dtype),
					hipMemcpyDeviceToHost,
					streams[0]));
		}

		// Count how many times were flagged
		assert(num_rescale_blocks >= 0);
		array4d_copy_to_host(&rescaler->nsamps); // must do this before updaing scaleoffset, which resets nsamps to zero
		tproc.start();

		for(int i = 0; i < nf*nbeams_in_total; ++i) {
			int nsamps = (int)rescaler->nsamps.d[i]; // nsamps is the number of unflagged samples from this block
			int nflagged = rescaler->sampnum - nsamps;
			// rescale.sampnum is the total number of samples that has gone into the rescaler since resetting
			assert (nflagged >= 0);
			num_flagged_times += nflagged;
		}

		// do rescaling if required
		if (num_rescale_blocks > 0 && blocknum % num_rescale_blocks == 0) {
			for(int iant = 0; iant < source->nants(); ++iant) {
				rescaler->update_scaleoffset(rescaler->options, iant);
			}

			// Count how many  channels have been flagged for this whole block
			// by looking at how many channels have scale==0
			array4d_copy_to_host(&rescaler->scale);
			for(int i = 0; i < nf*nbeams_in_total; ++i) {
				if (rescaler->scale.d[i] == 0) {
					// that channel will stay flagged for num_rescale_blocks
					num_flagged_beam_chans += num_rescale_blocks;
				}

				// it looks here like I'm counting twice, as we increment num_flagged_times outside the rescale_blocks_guard
				// but I'm a bit wary here, bcasue of teh danger, danger
//				// Count how many times have been flagged for this block
//				// TODO: DANGER DANGER! This doesn't count flagged times if num_rescale_blocks = 0
//				// This gave me a long headache at LAX when I set -s 1e30 stupidly.
//				int nsamps = (int)rescaler->nsamps.d[i];
//				// nsamps is the number of unflagged samples in nt*num_rescale_blocks samples
//				int nflagged = nt*num_rescale_blocks - nsamps;
//				assert (nflagged >= 0);
//				num_flagged_times += nflagged;
			}

			if (do_dump_rescaler) {
				dump_rescaler(iblock, rescaler);
			}
		}

		if (blocknum >= num_rescale_blocks) {
			/// Execute the FDMT
			fdmt_execute(&fdmt, rescale_buf.d_device, out_buf.d);
			if (dump_data) {
				dumparr("fdmt", iblock, &out_buf, false);
				dumparr("ostate", iblock, & fdmt.ostate, true);
			}
			//total_candidates += boxcar_threshonly(&out_buf, sampno, thresh, max_ncand_per_block, mindm, sink);
			tboxcar.start();
			boxcar_do_gpu (
					&fdmt.ostate,
					&boxcar_data,
					&boxcar_history,
					&boxcar_discards,
					thresh, max_ncand_per_block, mindm, maxbc, &candidate_list);
			tboxcar.stop();
			int ncand = candidate_list.copy_to_sink(sink);
			if (ncand >= max_ncand_per_block - 1) {
				num_candidate_overflow_blocks++;
			}
			total_candidates += ncand;
			if (dump_data) {
				dumparr("boxcar", iblock, &boxcar_data, true);
			}
		}
		tproc.stop();

		// release dada block from output -
		if (dada_sink != NULL) {
			gpuErrchk(hipStreamSynchronize(streams[0]));
			dada_sink->close_block();
		}

		blocknum++;
		iblock++;
	}

	tall.stop();

	// calculate array discards
	array4d_copy_to_host(&boxcar_discards);
	int total_discards = 0;
	for (int i = 0; i < array4d_size(&boxcar_discards); ++i) {
		total_discards += (int)boxcar_discards.d[i];
	}


	double boxcar_ngops = (double)nbeams_out*(double)nt*(double)nd*2.0*(double)NBOX/1e9;
	double data_nsecs = blocknum*nt*source->tsamp();

	double flagged_percent = ((double) num_flagged_beam_chans) / ((double) nf*nbeams_in_total*blocknum) * 100.0;
	double dm0_flagged_percent = ((double) num_flagged_times) / ((double) blocknum*nbeams_in_total*nt*nf) * 100.0;
	cout << " FREDDA Finished" << endl;
	cout << "Found " << total_candidates << " candidates" << endl;
	cout << "Discarded " << total_discards << " candidates for being too wide."<< endl;
	cout << num_candidate_overflow_blocks << " blocks overflowed the candidate buffer"<<endl;
	cout << "Processed " << blocknum << " blocks = "<< blocknum*nt << " samples = " << data_nsecs << " seconds" << " at " << data_nsecs/tall.wall_total()<< "x real time"<< endl;
	cout << "Freq auto-flagged " << num_flagged_beam_chans << "/" << (nf*nbeams_in_total*blocknum) << " channels = " << flagged_percent << "%" << endl;
	cout << "DM0 auto-flagged " << num_flagged_times << "/" << (blocknum*nbeams_in_total*nt*nf) << " samples = " << dm0_flagged_percent << "%" << endl;
	cout << "File reading " << endl << source->m_read_timer << endl;
	cout << "FREDDA Total "<< endl << tall << endl;
	cout << "FREDDA Procesing "<< endl << tproc << endl;
	cout << "Rescale "<< endl << trescale << endl;
	fdmt_print_timing(&fdmt);
	cout << "Boxcar "<< endl << tboxcar << endl;
	cout << "FDMT " << ((double)fdmt.nops)/1e9
			<< " Gops/iteration ran at: " << ((double)fdmt.nops) / (fdmt.t_iterations.get_average_time()/1e3)/1e9
			<< " GFLOPS" << endl;
	cout << "Boxcar " << boxcar_ngops
			<< " Gops/iteration. ran at: " << boxcar_ngops/(tboxcar.get_average_time()/1e3)
			<< " GFLOPS" << endl;
	struct rusage usage;
	getrusage(RUSAGE_SELF, &usage);
	cout << "Resources User: " << usage.ru_utime.tv_sec <<
			"s System:" << usage.ru_stime.tv_sec << "s MaxRSS:" << usage.ru_maxrss/1024/1024 << "MB" << endl;
	cout << "GPU Memory used " << (gpu_total_bytes - gpu_free_bytes)/1024/1024 << " of " << gpu_total_bytes /1024/124 << " MiB" << endl;
	delete source;
	if(dada_sink != NULL) {
		delete dada_sink;
	}
}

