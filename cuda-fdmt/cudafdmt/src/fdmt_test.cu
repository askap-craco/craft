#include "hip/hip_runtime.h"
//
//  fdmt_test.c
//  fdmt
//
//  Created by Keith Bannister on 19/07/2016.
//  Copyright (c) 2016 Keith Bannister. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "fdmt.h"
#include "cuda_fdmt.h"
#include "CudaTimer.h"

using namespace std;


int main(int argc, char* argv[])
{
    int nd = 512;
    int nt = 256;
    int nf = 512;
    int nbeams = 1;
    float fmax = 1440;
    float fmin = fmax - (float)nf;
    
    fdmt_dtype* din = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*nd*nf*nbeams);
    fdmt_dtype* dout = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*nd*nt*nbeams);
    printf("Starting!\n");
    
    if (argc != 3) {
        printf("Not enough arguments\n");
        exit(EXIT_FAILURE);
    }
    
    FILE* fin = fopen(argv[1], "r");
  
    //for(int i = 0; i < nd*nf; i++) {
    //din[i] = (fdmt_dtype)1;
    //}
    if (fin == NULL) {
        perror("Could not open input file");
        exit(EXIT_FAILURE);
    }
    fread(din, sizeof(fdmt_dtype), nd*nf, fin);

    // copy to all beams
    for(int b = 1; b < nbeams; b++) {
    	int idx = b*nd*nf;
    	memcpy(&din[idx], din, nd*nf*sizeof(fdmt_dtype));
    }
    fclose(fin);
    
    fdmt_t fdmt;
    fdmt_create(&fdmt, fmin, fmax, nf, nd, nt, nbeams);
    
    CudaTimer t;
    t.start();
    for(int i = 0; i < 1; i++) {
        fdmt_execute(&fdmt, din, dout);
    }
    t.stop();
    cout << "FDMT Execute loop took " << t << endl;


    
    FILE* fout = fopen(argv[2], "w");
    if (fout == NULL) {
        perror("Could not open output file");
        exit(EXIT_FAILURE);
    }
    fwrite(dout, sizeof(fdmt_dtype), nd*nf, fout);
    fclose(fout);
    printf("Wrote output file %s\n", argv[2]);
    
}
