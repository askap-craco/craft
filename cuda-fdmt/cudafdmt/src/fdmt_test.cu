#include "hip/hip_runtime.h"
//
//  fdmt_test.c
//  fdmt
//
//  Created by Keith Bannister on 19/07/2016.
//  Copyright (c) 2016 Keith Bannister. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <signal.h>
#include <limits.h>
#include <float.h>
#include <omp.h>
#include <sys/time.h>
#include <sys/resource.h>
#include "fdmt.h"
#include "array.h"
#include "boxcar.h"
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "DataSource.h"
#include "SigprocFile.h"
#include "SigprocFileSet.h"
#include "DataSource.h"
#include "DadaSource.h"
#include "DadaSet.h"
#include "FilDirSet.h"
#include "CandidateList.h"
#include "InvalidSourceFormat.h"
#include "Rescaler.h"
#include "rescale.h"
#include "DadaSink.h"
#include "FreddaParams.h"


using namespace std;


volatile bool stopped;

//typedef void (*sig_t) (int);

void handle_signal(int signal)
{
	stopped = true;
}

void dumparr(const char* prefix, const int blocknum, array4d_t* arr, bool copy=true)
{
	char fbuf[1024];
	sprintf(fbuf, "%s_e%d.dat", prefix, blocknum);
	if (copy) {
		array4d_copy_to_host(arr);
	}
	int nz = 0;
	int size = array4d_size(arr);
	for(int i = 0; i < size; i++) {
		if (arr->d[i] == 0.0) {
			nz += 1;
		}
	}

	//printf("Dumping %s %s %d zeros\n", prefix, fbuf, nz);
	array4d_dump(arr, fbuf);
}

void dump_rescaler(int iblock, Rescaler* rescaler)
{
	dumparr("mean", iblock, &rescaler->mean);
	dumparr("std", iblock, &rescaler->std);
	dumparr("kurt", iblock, &rescaler->kurt);
	dumparr("nsamps", iblock, &rescaler->nsamps);
	dumparr("dm0", iblock, &rescaler->dm0);
	dumparr("dm0count", iblock, &rescaler->dm0count);
	dumparr("dm0stats", iblock, &rescaler->dm0stats);
	dumparr("scale", iblock, &rescaler->scale);
	dumparr("offset", iblock, &rescaler->offset);
	dumparr("decay_offset", iblock, &rescaler->decay_offset);
}

int main(int argc, char* argv[])
{
	FreddaParams params; // this is new - I haven't finished refactoring everything to take advantage of FreddaParams
	params.parse(argc, argv);

	const int nt = params.nt;
	const int nd = params.nd;
	printf("\n");
	printf("Setting cuda device to %d\n", params.cuda_device);
	gpuErrchk( hipSetDevice(params.cuda_device));

	CpuTimer tall;
	CpuTimer tproc;
	CudaTimer trescale;
	CudaTimer tboxcar;
	CudaTimer tdump;
	tall.start();

	DataSource* source = NULL;
	DadaSet* dada_source = NULL; // for debugging
	try {
		// load sigproc file
		SigprocFileSet* fs_source = new SigprocFileSet(nt, params.argc, params.argv);
		source = fs_source;
	} catch (InvalidSourceFormat& e) {
		try {
			dada_source = new DadaSet(nt, params.argc, params.argv);
			source = dada_source;
		} catch (InvalidSourceFormat& e) {
			try {
				source = new FilDirSet(nt, params.argc, params.argv);
			} catch (InvalidSourceFormat& e) {
				printf("No valid inputs\n");
				exit(EXIT_FAILURE);
			}
		}
	}
	assert(source != NULL);
	int num_skip_blocks = params.seek_seconds / source->tsamp() / nt;
	printf("Seeking to start of data: block %d nsamples=%d time=%fs\n", num_skip_blocks, num_skip_blocks*nt, num_skip_blocks*nt*source->tsamp());
	if (num_skip_blocks > 0) {
		source->seek_sample(nt*num_skip_blocks);
	}

	// need to set source after seek sample otherwise start sample not correctly set
	params.set_source(*source);

	const int nf = params.nf; // Don't know nf until we know the source
	bool negdm = (params.nd < 0);
	CandidateSink sink(source, params.out_filename, negdm, params.udp_host, params.udp_port);
	cout << "spf tsamp " << source->tsamp()<< " ants " << source->nants() << " nbeams " << source->nbeams()
			<< " npols "<< source->npols() << " fch1 " << source->fch1() << " nchans "
			<< source->nchans() << " foff " << source->foff() << endl;

	printf("S/N Threshold %f Max ncand per block %d mindm %d \n", params.thresh, params.max_ncand_per_block, params.mindm);
	//rescale input buffer
	size_t in_buffer_bytes_per_ant = params.nbeams_per_antenna*nf*nt*params.nbits/8;
	uint8_t* in_buffer_device;
	printf("Copy in buffer size = %d MB per ant = %d MB TOTAL \n", in_buffer_bytes_per_ant/(1024l*1024l), in_buffer_bytes_per_ant*source->nants()/(1024l*1024l));
	gpuErrchk( hipMalloc((void**) &in_buffer_device, in_buffer_bytes_per_ant*source->nants() ));


	DadaSink* dada_sink = NULL;
	if (params.export_dada_key != -1) {
		char* hdr = NULL;
		if (dada_source != NULL) {
			hdr = dada_source->get_source_at(0)->get_header();
		}
		dada_sink = new DadaSink(params.export_dada_key, hdr, params);
	}

	const bool dump_data = params.dump_data;
	const int nbeams_out = params.nbeams_out;
	// rescale output buffer
	array4d_t rescale_buf;
	rescale_buf.nw = nbeams_out;
	rescale_buf.nx = nf;
	rescale_buf.ny = 1;
	rescale_buf.nz = nt;
	array4d_malloc(&rescale_buf, dump_data, true);

	// rescale junk buffer for first integration only - bleah
	array4d_t rescale_junk_buf;
	rescale_junk_buf.nw = nbeams_out;
	rescale_junk_buf.nx = nf;
	rescale_junk_buf.ny = 1;
	rescale_junk_buf.nz = nt;
	array4d_malloc(&rescale_junk_buf, false, true);

	// FDMT output buffer
	array4d_t out_buf;
	out_buf.nw = nbeams_out;
	out_buf.nx = 1;
	out_buf.ny = nd;
	out_buf.nz = nt;
	array4d_malloc(&out_buf, dump_data, true);


	// create rescaler
	RescaleOptions rescale = {};
	rescale.interval_samps = nt;
	rescale.target_mean = 0.0;
	rescale.target_stdev = 1.0/sqrt(params.nbeams_summed);
	rescale.decay_constant = 0.35 * params.decay_timescale / source->tsamp(); // This is how the_decimator.C does it, I think.
	rescale.mean_thresh = params.mean_thresh;
	rescale.std_thresh = params.std_thresh;
	rescale.kurt_thresh = params.kurt_thresh;
	rescale.flag_grow = params.flag_grow;
	rescale.dm0_thresh = params.dm0_thresh;
	rescale.cell_thresh = params.cell_thresh;
	rescale.gtest_thresh = params.gtest_thresh;
	rescale.invert_freq = (params.foff < 0);
	rescale.subtract_dm0 = params.subtract_dm0;
	rescale.nt = nt;
	rescale.nf = nf;
	rescale.nbeams_per_ant = params.nbeams_per_antenna;
	rescale.nants = source->nants();
	rescale.polsum = params.polsum;
	rescale.nbits = source->nbits();
	rescale.in_order = source->data_order();
	// set guess of initial scale and offset to dm0 thresholding works
	printf("Rescaling to mean=%f stdev=%f decay constant=%f mean/std/kurtosis/dm0/Cell/gtest thresholds: %0.1f/%0.1f/%0.1f/%0.1f/%0.1f/%0.1f grow flags by %d channels\n",
			rescale.target_mean,rescale.target_stdev,
			rescale.decay_constant,
			rescale.mean_thresh, rescale.std_thresh, rescale.kurt_thresh,
			rescale.dm0_thresh, rescale.cell_thresh, rescale.gtest_thresh,
			rescale.flag_grow);
	Rescaler* rescaler = new Rescaler(rescale, params);
	rescaler->set_scaleoffset(1.0f, 0.0f); // Just pass it straight through without rescaling

	// HACK! - ASKAP beam71 is useless CRAFT-25
	rescaler->flag_beam(71);

	// Create fdmt
	fdmt_t fdmt;
	printf("Creating FDMT fmin=%f fmax=%f nf=%d nd=%d nt=%d nbeams=%d nbeams_alloc=%d\n",
			params.fmin, params.fmax, nf, nd, nt, nbeams_out, params.nbeams_alloc);
	fdmt_create(&fdmt, params.fmin, params.fmax, nf, nd, nt, nbeams_out, params.nbeams_alloc, dump_data);

	int blocknum = 0;
	int iblock = num_skip_blocks;
	unsigned long long total_candidates = 0;
	unsigned long long num_candidate_overflow_blocks = 0;
	// make boxcar history
	array4d_t boxcar_history;
	boxcar_history.nw = 1;
	boxcar_history.nx = nbeams_out;
	boxcar_history.ny = nd;
	boxcar_history.nz = NBOX;
	array4d_malloc(&boxcar_history, dump_data, true);
	array4d_zero(&boxcar_history);
	// make boxcar discards
	array4d_t boxcar_discards;
	boxcar_discards.nw = 1;
	boxcar_discards.nx = 1;
	boxcar_discards.ny = nbeams_out;
	boxcar_discards.nz = nd;
	array4d_malloc(&boxcar_discards, true, true);
	array4d_cuda_memset(&boxcar_discards, 0);

	// make boxcar output.
	// TODO: Only allocate on GPU if we'll be dumping it to dis.
	// Otherwise, we'll just use candidate lists and save on a bucketload of memory
	array4d_t boxcar_data;
	boxcar_data.nw = nbeams_out;
	boxcar_data.nx = nd;
	boxcar_data.ny = nt;
	boxcar_data.nz = NBOX;
	array4d_malloc(&boxcar_data, dump_data, dump_data);
	array4d_zero(&boxcar_data);

	CandidateList candidate_list(params.max_ncand_per_block);

	// measure bytes used
	size_t gpu_free_bytes, gpu_total_bytes;
	gpuErrchk(hipMemGetInfo( &gpu_free_bytes, &gpu_total_bytes ));

	// add signal handler
	signal(SIGHUP, &handle_signal);
	signal(SIGINT, &handle_signal);
	signal(SIGTERM, &handle_signal);
	uint64_t num_flagged_beam_chans = 0;
	uint64_t num_flagged_times = 0;

	// Create streams - one for each antenan
	const int MAX_NANT = 72;
	hipStream_t streams[MAX_NANT];
	assert(source->nants() <= MAX_NANT);
	for (int i = 0; i < source->nants(); i++) {
		gpuErrchk(hipStreamCreate(&streams[i]));
		//streams[i] = 0;
	}

	while (true) {
		if (stopped) {
			printf("Stopped due to signal received\n");
			break;
		}
		if (blocknum >= params.max_nblocks) {
			printf("Max block exceeded. Quitting %d %d", blocknum, params.max_nblocks);
			break;
		}

		rescaler->reset_output(rescale_buf); // set output buffer to zero - each rescale update will add the result into the buffer

		fdmt.t_copy_in.start();

//#pragma omp parallel
		for(int iant = 0; iant < source->nants(); iant++) {
			// read samples from input - one antenna at a time.
			void* read_buf;
			int this_nt = source->read_samples_ant(&read_buf, iant);
			if (this_nt != nt) { // WE've run out of samples
				printf("iant %d returned %d samples but nt=%d. Finishing \n", iant, this_nt, nt);
				stopped = true;
				break;
			}

			uint8_t* this_ant_buffer = in_buffer_device + iant*in_buffer_bytes_per_ant;

			// Asynchronous copy goes onto the stream for that antenna - each antenna stream also has update and scaleoffset kernes
			gpuErrchk(hipMemcpyAsync(this_ant_buffer,
					read_buf, in_buffer_bytes_per_ant*sizeof(uint8_t), hipMemcpyHostToDevice, streams[iant]));

			if (blocknum == 0 && params.num_rescale_blocks > 0) { // if first block rescale and update with no
				// flagging so we can work out roughly what the scales are
				// Send output to junk buffer - silly but will fix later
				// TODO: Remove junk buffer to save memory
				rescaler->process_ant_block(rescale_junk_buf, this_ant_buffer, rescaler->noflag_options, iant, streams[iant]);

				// update scale and offset
				rescaler->update_scaleoffset(rescaler->noflag_options, iant, streams[iant]);

				// Reset rescale stats for this antenna only
				rescaler->reset_ant_stats_for_first_block(iant);

			}

			// this time we rescale with the flagging turned on
			rescaler->process_ant_block(rescale_buf, this_ant_buffer, rescaler->options, iant, streams[iant]);

		}
		gpuErrchk(hipDeviceSynchronize()); // Synchonize after doing all those asynchronous, multistream things
		rescaler->finish_all_ants();

		fdmt.t_copy_in.stop();

		if (stopped) {// if we've run out of samples
			printf("Run out of samples\n");
			break;
		}

		if (dump_data) {
			dumparr("inbuf", iblock, &rescale_buf);
		}

		// Do asynchronous copy to dada output using the copy stream for antenna 0
		if (dada_sink != NULL) {
			void* outptr = dada_sink->open_block();
			gpuErrchk(hipMemcpyAsync(outptr,
					rescale_buf.d_device,
					array4d_size(&rescale_buf)*sizeof(rescale_dtype),
					hipMemcpyDeviceToHost,
					streams[0]));
		}



		// do rescaling if required
		if (params.num_rescale_blocks > 0 && blocknum % params.num_rescale_blocks == params.num_rescale_blocks - 1) {
			rescaler->update_rescale_statistics();
		}

		if (blocknum >= params.num_rescale_blocks) {
			/// Execute the FDMT
			fdmt_execute(&fdmt, rescale_buf.d_device, out_buf.d);
			if (dump_data) {
				dumparr("fdmt", iblock, &out_buf, false);
				dumparr("ostate", iblock, & fdmt.ostate, true);
			}
			//total_candidates += boxcar_threshonly(&out_buf, sampno, thresh, max_ncand_per_block, mindm, sink);
			tboxcar.start();
			boxcar_do_gpu (
					&fdmt.ostate,
					&boxcar_data,
					&boxcar_history,
					&boxcar_discards,
					params.thresh, params.max_ncand_per_block, params.mindm, params.maxbc, &candidate_list);
			tboxcar.stop();
			int ncand = candidate_list.copy_to_sink(sink);
			if (ncand >= params.max_ncand_per_block - 1) {
				num_candidate_overflow_blocks++;
			}
			total_candidates += ncand;
			if (dump_data) {
				dumparr("boxcar", iblock, &boxcar_data, true);
			}
		}
		tproc.stop();

		// release dada block from output -
		if (dada_sink != NULL) {
			gpuErrchk(hipStreamSynchronize(streams[0]));
			dada_sink->close_block();
		}

		blocknum++;
		iblock++;
	}

	tall.stop();

	// calculate array discards
	array4d_copy_to_host(&boxcar_discards);
	int total_discards = 0;
	for (int i = 0; i < array4d_size(&boxcar_discards); ++i) {
		total_discards += (int)boxcar_discards.d[i];
	}

	const int nbeams_in_total = params.nbeams_in_total;
	double boxcar_ngops = (double)nbeams_out*(double)nt*(double)nd*2.0*(double)NBOX/1e9;
	double data_nsecs = blocknum*nt*source->tsamp();

	double flagged_percent = ((double) num_flagged_beam_chans) / ((double) nf*nbeams_in_total*blocknum) * 100.0;
	double dm0_flagged_percent = ((double) num_flagged_times) / ((double) blocknum*nbeams_in_total*nt*nf) * 100.0;
	cout << " FREDDA Finished" << endl;
	cout << "Found " << total_candidates << " candidates" << endl;
	cout << "Discarded " << total_discards << " candidates for being too wide."<< endl;
	cout << num_candidate_overflow_blocks << " blocks overflowed the candidate buffer"<<endl;
	cout << "Processed " << blocknum << " blocks = "<< blocknum*nt << " samples = " << data_nsecs << " seconds" << " at " << data_nsecs/tall.wall_total()<< "x real time"<< endl;
	cout << "Freq auto-flagged " << num_flagged_beam_chans << "/" << (nf*nbeams_in_total*blocknum) << " channels = " << flagged_percent << "%" << endl;
	cout << "DM0 auto-flagged " << num_flagged_times << "/" << (blocknum*nbeams_in_total*nt*nf) << " samples = " << dm0_flagged_percent << "%" << endl;
	cout << "File reading " << endl << source->m_read_timer << endl;
	cout << "FREDDA Total "<< endl << tall << endl;
	cout << "FREDDA Procesing "<< endl << tproc << endl;
	cout << "Rescale "<< endl << trescale << endl;
	cout << "Rescale dumping " << endl << tdump << endl;
	fdmt_print_timing(&fdmt);
	cout << "Boxcar "<< endl << tboxcar << endl;
	cout << "FDMT " << ((double)fdmt.nops)/1e9
			<< " Gops/iteration ran at: " << ((double)fdmt.nops) / (fdmt.t_iterations.get_average_time()/1e3)/1e9
			<< " GFLOPS" << endl;
	cout << "Boxcar " << boxcar_ngops
			<< " Gops/iteration. ran at: " << boxcar_ngops/(tboxcar.get_average_time()/1e3)
			<< " GFLOPS" << endl;
	struct rusage usage;
	getrusage(RUSAGE_SELF, &usage);
	cout << "Resources User: " << usage.ru_utime.tv_sec <<
			"s System:" << usage.ru_stime.tv_sec << "s MaxRSS:" << usage.ru_maxrss/1024/1024 << "MB" << endl;
	cout << "GPU Memory used " << (gpu_total_bytes - gpu_free_bytes)/1024/1024 << " of " << gpu_total_bytes /1024/124 << " MiB" << endl;
	delete source;
	delete rescaler;
	if(dada_sink != NULL) {
		delete dada_sink;
	}
}

