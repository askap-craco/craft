#include "hip/hip_runtime.h"
//
//  fdmt_test.c
//  fdmt
//
//  Created by Keith Bannister on 19/07/2016.
//  Copyright (c) 2016 Keith Bannister. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>
#include <sys/time.h>
#include <sys/resource.h>
//#include <omp.h>
#include "fdmt.h"
#include "array.h"
#include "boxcar.h"
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "DataSource.h"
#include "SigprocFile.h"
#include "SigprocFileSet.h"

#include "rescale.h"


using namespace std;

void runtest_usage() {
	fprintf(stderr,
			"fdmt_test [options] infile outfile\n"
			"	-d Number of dispersion trials\n"
			"	-t Samples per block\n"
			"	-f Number of frequency channels\n"
			"	-b Number of beams\n"
			"	-x Maximum frequency (MHz)\n"
			"	-h Print this message\n"
	);
	exit(EXIT_FAILURE);
}

int main(int argc, char* argv[])
{
	printf("Test!");
	int nd = 512;
	int nt = 256;
	int num_skip_blocks = 4;
	int num_rescale_blocks = 2;
	float decay_timescale = 0.2; // Seconds?
	char ch;
	float thresh = 10.0;
	const char* out_filename = "fredda.cand";
	bool dump_data = false;
	int cuda_device = 0;
	CpuTimer tall;
	CpuTimer trescale;
	CpuTimer tboxcar;

	tall.start();
	while ((ch = getopt(argc, argv, "d:t:s:o:x:r:S:D:g:h")) != -1) {
		switch (ch) {
		case 'd':
			nd = atoi(optarg);
			break;
		case 't':
			nt = atoi(optarg);
			break;
		case 's':
			decay_timescale = atof(optarg);
			break;
		case 'o':
			out_filename = optarg;
			break;
		case 'x':
			thresh = atof(optarg);
			break;
		case 'D':
			dump_data = true;
			break;
		case 'r':
			num_rescale_blocks = atoi(optarg);
			break;
		case 'S':
			num_skip_blocks = atoi(optarg);
			break;
		case 'g':
			cuda_device = atoi(optarg);
			break;
		case '?':
		case 'h':
		default:
			runtest_usage();
		}
	}
	argc -= optind;
	argv += optind;

	if (argc == 0) {
		printf("Not enough arguments\n");
		exit(EXIT_FAILURE);
	}

	printf("Setting cuda device to %d\n", cuda_device);
	gpuErrchk( hipSetDevice(cuda_device));

	// Load sigproc file
	SigprocFileSet source(argc, argv);

	CandidateSink sink(&source, out_filename);
	cout << "spf tsamp " << source.tsamp()<< " nbeams " << source.nbeams() << " fch1 " << source.fch1() << " nchans "
			<< source.nchans() << "foff " << source.foff() << endl;
	int nbeams = source.nbeams();
	int nf = source.nchans();
	size_t in_chunk_size = nbeams*nf*nt;

	// Create read buffer
	uint8_t* read_buf = (uint8_t*) malloc(sizeof(uint8_t) * in_chunk_size);
	array4d_t read_arr;
	read_arr.nw = 1;
	read_arr.nx = nt;
	read_arr.ny = nbeams;
	read_arr.nz = nf;
	assert(read_buf);


	array4d_t rescale_buf;
	rescale_buf.nw = nbeams;
	rescale_buf.nx = nf;
	rescale_buf.ny = 1;
	rescale_buf.nz = nt;
	array4d_malloc_hostonly(&rescale_buf);

	array4d_t out_buf;
	out_buf.nw = nbeams;
	out_buf.nx = 1;
	out_buf.ny = nd;
	out_buf.nz = nt;
	array4d_malloc_hostonly(&out_buf);

	// create rescaler
	rescale_t rescale;
	rescale.interval_samps = nt;
	rescale.target_mean = 0.0;
	rescale.target_stdev = 1.0/sqrt((float) nf);
	rescale.decay_constant = 0.35 * decay_timescale / source.tsamp(); // This is how the_decimator.C does it, I think.
	printf("Rescaling to mean=%f stdev=%f decay constant=%f\n",rescale.target_mean,rescale.target_stdev, rescale.decay_constant);
	rescale_allocate(&rescale, nbeams*nf);

	float foff =  (float) source.foff();
	assert(foff < 0);
	float fmax = (float) source.fch1() - foff; // The FDMT seems to want this to make sense of the world. Not sure why.
	float fmin = fmax + nf*foff;
	fdmt_t fdmt;
	printf("Creating FDMT fmin=%f fmax=%f nf=%d nd=%d nt=%d nbeams=%d\n", fmin, fmax, nf, nd, nt, nbeams);
	fdmt_create(&fdmt, fmin, fmax, nf, nd, nt, nbeams);
	printf("Seeking to start of data: nblocks=%d nsamples=%d time=%fs\n", num_skip_blocks, num_skip_blocks*nt, num_skip_blocks*nt*source.tsamp());
	source.seek_sample(num_skip_blocks*nt);
	int blocknum = 0;

	while (source.read_samples_uint8(nt, read_buf) == nt) {
		//size_t nt2 = fin.read_samples_uint8(nt, read_buf2);
		//assert(nt2 = nt);
		// File is in TBF order
		// Output needs to be BFT order
		// Do transpose and cast to float on the way through
		// TODO: Optimisation: cast to float and do rescaling in SIMD

		trescale.start();
		#pragma omp parallel for
		for(int t = 0; t < nt; ++t) {
			#pragma omp parallel for
			for (int b = 0; b < nbeams; ++b) {
				for (int f = 0; f < nf; ++f) {
					// NOTE: FDMT expects channel[0] at fmin
					// so invert the frequency axis if the frequency offset is negative
					int outf = f;
					if (foff < 0) {
						outf = nf - f - 1;
					}
					int inidx = array4d_idx(&read_arr, 0, b, t, f);
					int outidx = array4d_idx(&rescale_buf, b, outf, 0, t);

					//printf("t=%d b=%d f=%d inidx=%d outidx=%d\n", t, b, f, inidx, outidx);
					// writes to inbuf
					size_t rs_idx = outf + nf*b;
					float v_rescale;
					//printf("Rescaling to mean=%f stdev=%f decay constant=%f\n",rescale.target_mean,rescale.target_stdev, rescale.decay_constant);

					v_rescale = rescale_update_decay_float_single(&rescale, rs_idx, (float) read_buf[inidx]);
					rescale_buf.d[outidx] = v_rescale;
					//printf("block=%d t=%d b=%d f=%d vin=%d vout=%f \n", blocknum, t, b, f, read_buf[inidx], v_rescale);

				}
			}
		}
		trescale.stop();
		rescale.sampnum += nt; // WARNING: Need to do this because we're calling rescale*single. THink harder about how to do this beter

		char fbuf[1024];
		if (dump_data) {
			sprintf(fbuf, "inbuf_e%d.dat", blocknum);
			array4d_dump(&rescale_buf, fbuf);
			printf("Dumping input buffer %s\n", fbuf);
		}

		assert(num_rescale_blocks > 0);

		if (blocknum % num_rescale_blocks == 0) {
			rescale_update_scaleoffset(&rescale);
		}

		if (blocknum > num_rescale_blocks) {
			fdmt_execute(&fdmt, rescale_buf.d, out_buf.d);
			if (dump_data) {
				sprintf(fbuf, "fdmt_e%d.dat", blocknum);
				printf("Dumping fdmt buffer %s\n", fbuf);
				array4d_dump(&out_buf, fbuf);
			}
			tboxcar.start();
			boxcar_threshonly(&out_buf, thresh, sink);
			tboxcar.stop();

		}

		blocknum++;
	}

	printf("FREDDA Finished\n");
	tall.stop();
	cout << "FREDDA CPU "<< tall << endl;
	cout << "Rescale CPU "<< trescale << endl;
	cout << "Boxcar CPU "<< tboxcar << endl;
	cout << "File reading " << source.read_timer << endl;
	fdmt_print_timing(&fdmt);

	struct rusage usage;
	getrusage(RUSAGE_SELF, &usage);
	cout << "Resources User: " << usage.ru_utime.tv_sec <<
			"s System:" << usage.ru_stime.tv_sec << "s MaxRSS:" << usage.ru_maxrss/1024/1024 << "MB" << endl;
}
int runtest(int argc, char* argv[])
{
	int nd = 512;
	int nt = 256;
	int nf = 336;
	int nbeams = 1;
	float fmax = 1440;
	char ch;
	while ((ch = getopt(argc, argv, "d:t:f:b:x:g:h")) != -1) {
		switch (ch) {
		case 'd':
			nd = atoi(optarg);
			break;
		case 't':
			nt = atoi(optarg);
			break;
		case 'f':
			nf = atoi(optarg);
			break;
		case 'b':
			nbeams = atoi(optarg);
			break;
		case 'x':
			fmax = atof(optarg);
			break;
		case '?':
		case 'h':
		default:
			runtest_usage();
		}
	}
	argc -= optind;
	argv += optind;

	float fmin = fmax - (float)nf;

	int blockin = nf*nt;
	int blockout = nd*nt;
	fdmt_dtype* din = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockin*nbeams);
	fdmt_dtype* din_tmp = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockin*nbeams);
	fdmt_dtype* dout = (fdmt_dtype*) malloc(sizeof(fdmt_dtype)*blockout*nbeams);
	printf("Starting! fmin=%f fmax=%f nbeams=%d nf=%d nd=%d nt=%d\n", fmin, fmax, nbeams, nf, nd, nt);

	if (argc != 2) {
		printf("Not enough arguments\n");
		exit(EXIT_FAILURE);
	}

	FILE* fin = fopen(argv[0], "r");
	if (fin == NULL) {
		perror("Could not open input file");
		exit(EXIT_FAILURE);
	}

	FILE* fout = fopen(argv[1], "w");
	if (fout == NULL) {
		perror("Could not open output file");
		exit(EXIT_FAILURE);
	}


	fdmt_t fdmt;
	fdmt_create(&fdmt, fmin, fmax, nf, nd, nt, nbeams);

	int nbox = 32;
	array4d_t boxout;
	boxout.nw = nbeams;
	boxout.nx = nd;
	boxout.ny = nt;
	boxout.nz = nbox;
	array4d_malloc(&boxout);

	// read input file until exhausted
	while (fread(din_tmp, sizeof(fdmt_dtype), blockin, fin) == blockin) {

		// File is in TF format. We need FT order.
		// Do the transpose
		for(int t = 0; t < nt; ++t) {
			for (int f = 0; f < nf; f++) {
				din[f*nt + t] = din_tmp[f + nf*t];
			}
		}
		// copy to all beams
		for(int b = 1; b < nbeams; b++) {
			int idx = b*blockin;
			//memcpy(&din[idx], din, blockin*sizeof(fdmt_dtype));
		}

		CudaTimer t;
		t.start();
		for(int i = 0; i < 1; i++) {
			fdmt_execute(&fdmt, din, dout);
		}

		boxcar_do(&fdmt.states[fdmt.curr_state_idx], &boxout);

		t.stop();
		cout << "FDMT Execute loop took " << t << endl;
		fwrite(dout, sizeof(fdmt_dtype), blockout, fout);
		cout << "Wrote " << blockout << " elements to outfile. First two are:" << dout[0] << dout[1] << endl;
	}
	fclose(fin);
	fclose(fout);
}
