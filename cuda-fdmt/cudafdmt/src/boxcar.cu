#include "hip/hip_runtime.h"
/*
 * Boxcar functions
 */

#include "fdmt_utils.h"
#include "array.h"
#include "CandidateSink.h"
#include "boxcar.h"

int mod(int a, int b)
{
    int r = a % b;
    return r < 0 ? r + b : r;
}

__global__ void boxcar_do_kernel(const __restrict__ fdmt_dtype* indata,
		fdmt_dtype* __restrict__ outdata,
		int nt)
{
	__shared__ fdmt_dtype history[NBOX];
	int ibeam = blockIdx.x;
	int nbeams = gridDim.x;

	int idt = blockIdx.y;
	int max_dt = gridDim.y;

	int off = max_dt*(idt + ibeam*nbeams);
	const fdmt_dtype* iptr = indata + off;
	fdmt_dtype* optr = outdata + off;

	int ibc = threadIdx.x;
	int tidx = threadIdx.x;

	// initialise history
	// TODO: Load history from previous run. This will be overwritten with the state
	//history[ibc] = iptr[ibc];
	history[ibc] = 0;

	// Initialise state from history. This is basically a 'sum scan' in reverse
	// order. i.e. history[n] = sum_{i=n+1}^{NBOX}{history[i]}. Ideally you'd do a
	// Work efficient parallel scan (see http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html)
	// But, given we're only doing 32 sums, it's probably overkill for what we want. Basically we'll
	// Sum in place in the history, and then set the thread states once we're done
	if (ibc == 0) {
		for(int t = NBOX-2; t >= 0; --t) {
			history[t] += history[t+1];
		}
	}

	__syncthreads();
	// setup the state for *this* thread (which sits in a register)
	fdmt_dtype state = history[ibc];

	// Need to load the history into shared memory again
	history[ibc] = iptr[ibc];

	for(int t = 0; t < nt; ++t) {
		// Should be a LDU instruction - global load across all threads
		fdmt_dtype v = iptr[t];
		int history_index = (t - ibc - 1) % NBOX;
		// the access to the history should have no bank conflicts, as each thread access a different bank
		state += v - history[history_index];

		// write input back to history
		if (ibc == 0) {
			history[history_index] = v;
		}

		// write state into output
		optr[ibc] = state;

		// increment output pointer
		optr += NBOX;

		__syncthreads();
	}

	// TODO: write history so we can do previous run

}

int boxcar_do_cpu(const array4d_t* indata, array4d_t* outdata, array4d_t* boxcar_history)
{
	// Inshape: [nbeams, 1, ndt, nt]
	// outshape: [nbeams, ndt, nt, nbox=32]
	// KB checked this code on 10 Dec 2016 late at night with much pain - and it works.

	int nbeams = indata->nw;
	assert(indata->nx == 1);
	int ndt = indata->ny;
	int nt = indata->nz;
	outdata->nw = nbeams;
	outdata->nx = ndt;
	outdata->ny = nt;
	outdata->nz = NBOX;

	//boxcar_history.nw = 1;
	//boxcar_history.nx = nbeams;
	//boxcar_history.ny = nd;
	//boxcar_history.nz = NBOX;

	fdmt_dtype* inp = indata->d;
	fdmt_dtype* outp = outdata->d;

	for(int b = 0; b < nbeams; ++b) {
		for(int idt = 0; idt < ndt; ++idt) {
			// initialise state from boxcar history
			fdmt_dtype state[NBOX];
			int histidx = array4d_idx(boxcar_history, 0, b, idt, 0);
			fdmt_dtype* history = &boxcar_history->d[histidx];

			// history increases to the left
			state[0] = history[0];
			for(int ibc = 1; ibc < NBOX; ++ibc) {
				state[ibc] = state[ibc-1] + history[ibc];
			}

			assert(state[0] == history[0]);

			for(int t = 0; t < nt; ++t) {
				int inidx = array4d_idx(indata, b, 0, idt, t);
				fdmt_dtype vin = inp[inidx];
				for (int ibc = 0; ibc < NBOX; ++ibc) {
					int history_index = mod((-t + ibc),  NBOX);
					int outidx = array4d_idx(outdata, b, idt, t, ibc);
					state[ibc] += vin - history[history_index];
					outp[outidx] = state[ibc]/(sqrtf((float) (ibc + 1)));
				}
				int ohistidx = mod(-t-1, NBOX);
				history[ohistidx] = vin;
			}
		}
	}
}
int boxcar_do(array4d_t* indata, array4d_t* outdata)
{
	// Inshape: [nbeams, 1, ndt, nt]
	// outshape: [nbeams, ndt, nt, nbox=32]
	return 0;

}

int boxcar_threshonly(const array4d_t* indata, size_t sampno, fdmt_dtype thresh, int max_ncand_per_block, int mindm,
		CandidateSink& sink) {
	int nbeams = indata->nw;
	assert(indata->nx == 1);
	int ndt = indata->ny;
	int nt = indata->nz;
	int ncand = 0;

#pragma omp parallel for shared(ncand)
	for(int b = 0; b < nbeams; ++b) {
		for(int idt = mindm; idt < ndt; ++idt) {
			int off = array4d_idx(indata, b, 0, idt, 0);
			for(int t = 0; t < nt; ++t) {
				int inidx = off + t;
				fdmt_dtype v = indata->d[inidx];
				if (v > thresh && ncand < max_ncand_per_block) {
#pragma omp critical
					{
						sink.add_candidate(b, idt, t+sampno, 0, v);
						ncand += 1;
					}
				}
			}
		}
		if (ncand >= max_ncand_per_block) {

		}
	}

	return ncand;
}
