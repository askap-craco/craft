#include "hip/hip_runtime.h"
/*
 * Rescaling utilities
 * Author: Keith Bannister <keith.bannister@csiro.au>
 */

#include "rescale.h"
#include <stdint.h>
#include <unistd.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include "array.h"

typedef float rescale_dtype;

void* rescale_malloc(size_t sz)
{
	void* ptr = malloc(sz);
	assert(ptr);
	return ptr;
}

void rescale_update_scaleoffset(rescale_t* rescale)
{
	//assert(rescale->interval_samps >= 0);
	assert(rescale->target_stdev > 0);
	float nsamp = (float) rescale->sampnum;
	for (unsigned i = 0; i < rescale->num_elements; i++) {
		float mean = rescale->sum[i]/nsamp;
		float meansq = rescale->sum2[i]/nsamp;
		float variance = meansq - mean*mean;

		if (rescale->interval_samps == 0) { // Don't do rescaling
			rescale->scale[i] = 1.0;
			rescale->offset[i] = 0.0;
		} else {

			if (variance == 0.0) {
				rescale->scale[i] = rescale->target_stdev;
			} else {
				rescale->scale[i] = rescale->target_stdev / sqrt(variance);
			}

			rescale->offset[i] = -mean + rescale->target_mean/rescale->scale[i];
		}

		// reset values to zero
		rescale->sum[i] = 0.0;
		rescale->sum2[i] = 0.0;
	}

	rescale->sampnum = 0;
}

void rescale_update_none(rescale_t* rescale, float* inx, float*outx) 
{
	for (unsigned i = 0; i <  rescale->num_elements; i++) {
		float vin = inx[i];
		outx[i] = vin;
	}
}


void rescale_update_float(rescale_t* rescale, float* fdata, float* sampbuf, unsigned istart)
{
	float* inx = &fdata[istart];
	float* outx = &sampbuf[istart];
	for (unsigned i = 0; i <  rescale->num_elements; i++) {
		float vin = inx[i];
		float vin2 = vin*vin;
		rescale->sum[i] += vin;
		rescale->sum2[i] += vin2;
		outx[i] = (vin + rescale->offset[i]) * rescale->scale[i];
	}
}

void rescale_update_float_polsum(rescale_t* rescale, float* fdata, float* sampbuf, unsigned istart)
{
	float* inx = &fdata[istart];
	float* outx = &sampbuf[istart];

	for (unsigned i = 0; i < rescale->num_elements/2; i++) {
		unsigned j = 2*i;
		float vin = inx[j];
		rescale->sum[j] += vin;
		rescale->sum2[j] += vin*vin;

		float uin = inx[j+1];
		rescale->sum[j+1] += uin;
		rescale->sum2[j+1] += uin*uin;

		float vscale = (vin + rescale->offset[j]) * rescale->scale[j];
		float uscale = (uin + rescale->offset[j+1]) * rescale->scale[j+1];

		float vout = (vscale + uscale)/2.0;

		outx[j] = vout;
	}
}

void rescale_update_uint8(rescale_t* rescale, float* fdata, uint8_t* sampbuf, unsigned istart)
{
	float* in = &fdata[istart];
	uint8_t* out = &sampbuf[istart];

	for (unsigned i = 0; i < rescale->num_elements; i++) {
		float vin = in[i];
		rescale->sum[i] += vin;
		rescale->sum2[i] += vin*vin;
		float vout = (vin + rescale->offset[i]) * rescale->scale[i];

		if (vout < 0) {
			out[i] = 0;
		} else if (vout > 255) {
			out[i] = 255;
		} else {
			out[i] = (uint8_t) vout;
		}
	}
}

void rescale_update_uint8_polsum(rescale_t* rescale, float* fdata, uint8_t* sampbuf, unsigned istart)
{
	float* in = &fdata[istart];
	uint8_t* out = &sampbuf[istart];

	for (unsigned i = 0; i < rescale->num_elements/2; i++) {
		unsigned j=2*i;
		float vin = in[j];
		rescale->sum[j] += vin;
		rescale->sum2[j] += vin*vin;

		float uin=in[j+1];
		rescale->sum[j+1] += uin;
		rescale->sum2[j+1] += uin*uin;

		float vscale = (vin + rescale->offset[j]) * rescale->scale[j];
		float uscale = (uin + rescale->offset[j+1]) * rescale->scale[j+1];

		float vout = (vscale+uscale)/2.0;
		if (vout < 0) {
			out[j] = 0;
		} else if (vout > 255) {
			out[j] = 255;
		} else {
			out[j] = (uint8_t) vout;
		}
	}
}

void rescale_update_int8(rescale_t* rescale, float* __restrict__ in, int8_t* __restrict__ out)
{

	for (unsigned i = 0; i < rescale->num_elements; i++) {
		float vin = in[i];
		rescale->sum[i] += vin;
		rescale->sum2[i] += vin*vin;
		float vout = (vin + rescale->offset[i]) * rescale->scale[i];
		if (vout < -128) {
			out[i] = -128;
		} else if (vout > 127) {
			out[i] = 127;
		} else {
			out[i] = (int8_t) vout;
		}
	}
	rescale->sampnum++;
	if (rescale->sampnum >= rescale->interval_samps) {
		rescale_update_scaleoffset(rescale);
	}

}

float rescale_update_decay_float_single(rescale_t* rescale, uint64_t i, float vin)
{
	assert(i < rescale->num_elements);
	float k = rescale->decay_constant;
	assert(k >= 0);

	rescale->sum[i] += vin;
	rescale->sum2[i] += vin*vin;
	float vout = (vin + rescale->offset[i]) * rescale->scale[i];
	rescale->decay_offset[i] = (vout + rescale->decay_offset[i]*k) / (1.0 + k);

	float out = vout - rescale->decay_offset[i];

	return out;
}

void rescale_update_decay_float(rescale_t* rescale, float* __restrict__ in, float* __restrict__ out)
{
	float k = rescale->decay_constant;

	for (unsigned i = 0; i < rescale->num_elements; i++) {
		float vin = in[i];
		rescale->sum[i] += vin;
		rescale->sum2[i] += vin*vin;
		float vout = (vin + rescale->offset[i]) * rescale->scale[i];
		rescale->decay_offset[i] = (vout + rescale->decay_offset[i]*k) / (1.0 + k);
		out[i] = vout - rescale->decay_offset[i];
	}
	rescale->sampnum++;
	if (rescale->sampnum >= rescale->interval_samps) {
		rescale_update_scaleoffset(rescale);
	}

}

void rescale_update_decay_uint8(rescale_t* rescale,  float* in, uint8_t* out)
{
	float k = rescale->decay_constant;

	for (unsigned i = 0; i < rescale->num_elements; i++) {
		float vin = in[i];
		rescale->sum[i] += vin;
		rescale->sum2[i] += vin*vin;
		float vout = (vin + rescale->offset[i]) * rescale->scale[i];
		rescale->decay_offset[i] = (vout + rescale->decay_offset[i]*k) / (1.0 + k);
		float rout = (vout - rescale->decay_offset[i]);
		if (rout < 0) {
			out[i] = 0;
		} else if (rout > 255) {
			out[i] = 255;
		} else {
			out[i] = (uint8_t) rout;
		}

	}

	rescale->sampnum++;
	if (rescale->sampnum >= rescale->interval_samps) {
		rescale_update_scaleoffset(rescale);
	}
}

rescale_t* rescale_allocate(rescale_t* rescale, uint64_t nelements) 
{
	size_t sz = nelements*sizeof(float);

	rescale->sum = (float*) rescale_malloc(sz);
	rescale->sum2 = (float*) rescale_malloc(sz);
	rescale->scale = (float*) rescale_malloc(sz);
	rescale->offset = (float*) rescale_malloc(sz);
	rescale->decay_offset = (float*) rescale_malloc(sz);
	rescale->sampnum = 0;
	rescale->num_elements = nelements;

	for(uint64_t i = 0; i < nelements; ++i) {
		rescale->sum[i] = 0;
		rescale->sum2[i] = 0;
		rescale->scale[i] = 1.0;
		rescale->offset[i] = 0.0;
		rescale->decay_offset[i] = 0.0;
	}

	//rescale_update_scaleoffset(rescale);
	return rescale;

}

rescale_dtype* rescale_cumalloc(uint64_t sz)
{
	rescale_dtype* ptr;
	gpuErrchk(hipMalloc((void**) &ptr, sz));
	gpuErrchk(hipMemset(ptr, 0, sz));
	return ptr;
}

void rescale_arraymalloc(array4d_t* arr, uint64_t nbeams, uint64_t nf, bool alloc_host)
{
	arr->nw = 1;
	arr->nx = 1;
	arr->ny = nbeams;
	arr->nz = nf;
	array4d_malloc(arr, alloc_host, true);
	array4d_zero(arr);
}

rescale_gpu_t* rescale_allocate_gpu(rescale_gpu_t* rescale, uint64_t nbeams, uint64_t nf, uint64_t nt, bool alloc_host)
{
	uint64_t nelements = nbeams*nf;
	size_t sz = nelements*sizeof(rescale_dtype);
	rescale_arraymalloc(&rescale->sum, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->sum2, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->sum3, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->sum4, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->mean, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->std, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->kurt, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->dm0, nbeams, nt, alloc_host);
	rescale_arraymalloc(&rescale->dm0count, nbeams, nt, alloc_host);
	rescale_arraymalloc(&rescale->dm0stats, nbeams, 4, alloc_host); // max, min, mean, var
	rescale_arraymalloc(&rescale->nsamps, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->scale, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->offset, nbeams, nf, alloc_host);
	rescale_arraymalloc(&rescale->decay_offset, nbeams, nf, alloc_host);
	array4d_set(&rescale->scale, 1.0);

	rescale->sampnum = 0;
	rescale->num_elements = nelements;
	rescale->nf = nf;
	rescale->nt = nt;
	rescale->nbeams = nbeams;

	return rescale;

}

void rescale_set_scale_offset_gpu(rescale_gpu_t* rescale, float scale, float offset)
{
	array4d_set(&rescale->scale, scale);
	array4d_set(&rescale->offset, offset);
}



void rescale_update_and_transpose_float(rescale_t& rescale, array4d_t& read_arr, array4d_t& rescale_buf, uint8_t* read_buf, bool invert_freq)
{
	int nbeams = rescale_buf.nw;
	int nf = rescale_buf.nx;
	int nt = rescale_buf.nz;
	assert(rescale_buf.ny == 1);
	rescale.sampnum += nt;
	for(int t = 0; t < nt; ++t) {
#pragma omp parallel for
		for (int b = 0; b < nbeams; ++b) {
			int instart = array4d_idx(&read_arr, 0, b, t, 0);

			for (int f = 0; f < nf; ++f) {
				// NOTE: FDMT expects channel[0] at fmin
				// so invert the frequency axis if the frequency offset is negative
				int outf = f;
				if (invert_freq) {
					outf = nf - f - 1;
				}
				int inidx = instart + f;
				int outidx = array4d_idx(&rescale_buf, b, outf, 0, t);

				//printf("t=%d b=%d f=%d inidx=%d outidx=%d\n", t, b, f, inidx, outidx);
				// writes to inbuf
				size_t rs_idx = outf + nf*b;
				float v_rescale;
				//printf("Rescaling to mean=%f stdev=%f decay constant=%f\n",rescale.target_mean,rescale.target_stdev, rescale.decay_constant);

				v_rescale = rescale_update_decay_float_single(&rescale, rs_idx, (float) read_buf[inidx]);
				rescale_buf.d[outidx] = v_rescale;
				//printf("block=%d t=%d b=%d f=%d vin=%d vout=%f \n", blocknum, t, b, f, read_buf[inidx], v_rescale);

			}
		}
	}
}


__global__ void rescale_calc_dm0_kernel (
		const uint8_t* __restrict__ inarr,
		const rescale_dtype* __restrict__ offsetarr,
		const rescale_dtype* __restrict__ scalearr,
		rescale_dtype* __restrict__ dm0arr,
		rescale_dtype* __restrict__ dm0count,
		int nf,
		int nt,
		rescale_dtype cell_thresh)
{
	// input = BTF order
	// dm0 order: BT
	// Rescale: BF order

	int ibeam = blockIdx.x;
	for(int t = threadIdx.x; t < nt; t += blockDim.x) {
		rescale_dtype dm0sum = 0.0;
		int nsamp = 0;
		for (int c = 0; c < nf; ++c) {
			int rsidx = c + nf*ibeam; // rescale index BF order
			// all these reads are nice and coalesced
			rescale_dtype offset = offsetarr[rsidx]; // read from global
			rescale_dtype scale = scalearr[rsidx]; // read from global
			int inidx = c + nf*(t + nt*ibeam); // input index : BTF order

			// coalesced read from global
			rescale_dtype vin = (rescale_dtype)inarr[inidx]; // read from global
			rescale_dtype vout = (vin + offset) * scale;
			if (fabs(vout) < cell_thresh && scale != 0.0f) {
				dm0sum += vout;
				++nsamp;
			}
		}

		int dm0idx = t + nt*ibeam;
		rescale_dtype correction = rsqrtf((float) nsamp);
		//dm0arr[dm0idx] = dm0sum * correction;
		dm0arr[dm0idx] = dm0sum;
		dm0count[dm0idx] = (float)nsamp;
	}
}

__global__ void rescale_calc_dm0stats_kernel (
		const rescale_dtype* __restrict__ dm0arr,
		const rescale_dtype* __restrict__ dm0countarr,
		rescale_dtype* __restrict__ dm0statarr,
		int nt)
{
	// dm0 order: BT
	// dm0stats order: BX
	// X is max,min,mean,var

	int ibeam = threadIdx.x;
	rescale_dtype dm0sum = 0.0;
	rescale_dtype dm0sum2 = 0.0;
	rescale_dtype nsampinit = dm0countarr[ibeam];
	rescale_dtype vinit =  dm0arr[ibeam] * rsqrtf(nsampinit); // normalise to sqrt number of additions
	rescale_dtype dm0min = vinit;
	rescale_dtype dm0max = vinit;


	for (int t = 0; t < nt; ++t) {
		int dmidx = t + nt*ibeam;
		rescale_dtype nsamp = dm0countarr[dmidx];
		rescale_dtype v = dm0arr[dmidx] * rsqrtf(nsamp); // normalise to sqrt number of additions

		dm0sum += v;
		dm0sum2 += v*v;
		if (v < dm0min) {
			dm0min = v;
		}
		if (v > dm0max) {
			dm0max = v;
		}

	}
	//dm0sumarr[ibeam] = dm0sum/((float) nt);
	rescale_dtype nsamp = (float) nt;
	rescale_dtype dm0mean = dm0sum/nsamp;
	rescale_dtype mean2 = dm0sum2/nsamp;
	rescale_dtype dm0var = mean2 - dm0mean*dm0mean;

	dm0statarr[ibeam + 0] = dm0max;
	dm0statarr[ibeam + 1] = dm0min;
	dm0statarr[ibeam + 2] = dm0mean;
	dm0statarr[ibeam + 3] = dm0var;

	//printf("DM stats ibeam=%d max/min/mean/var %f/%f/%f/%f\n", ibeam, dm0max, dm0min, dm0mean, dm0var);
}


__global__ void rescale_update_and_transpose_float_kernel (
		const uint8_t* __restrict__ inarr,
		rescale_dtype* __restrict__ sumarr,
		rescale_dtype* __restrict__ sum2arr,
		rescale_dtype* __restrict__ sum3arr,
		rescale_dtype* __restrict__ sum4arr,
		rescale_dtype* __restrict__ decay_offsetarr,
		rescale_dtype* __restrict__ nsampsarr,
		const rescale_dtype* __restrict__ offsetarr,
		const rescale_dtype* __restrict__ scalearr,
		const rescale_dtype* __restrict__ dm0arr,
		const rescale_dtype* __restrict__ dm0countarr,
		const rescale_dtype* __restrict__ dm0statarr,
		rescale_dtype* __restrict__ outarr,
		float decay_constant,
		float dm0_thresh,
		float cell_thresh,
		int nt,
		bool invert_freq,
		bool subtract_dm0)
{
	int ibeam = blockIdx.x;
	int c = threadIdx.x;
	int nf = blockDim.x;
	const rescale_dtype k = decay_constant;


	// input = BTF order
	// output = BFT order
	// Rescale order: BF
	// dm0 order: BT
	// dm0sum order: B
	// nsamps order: BF

	int rsidx = c + nf*ibeam; // rescale index: BF order
	// all these reads are nice and coalesced
	rescale_dtype sum = sumarr[rsidx]; // read from global memory
	rescale_dtype sum2 = sum2arr[rsidx]; // read from global
	rescale_dtype sum3 = sum3arr[rsidx];
	rescale_dtype sum4 = sum4arr[rsidx];
	rescale_dtype decay_offset = decay_offsetarr[rsidx];  // read from global
	rescale_dtype offset = offsetarr[rsidx]; // read from global
	rescale_dtype scale = scalearr[rsidx]; // read from global
	rescale_dtype nsamps = nsampsarr[rsidx]; // read from global

	int outc;
	if (invert_freq) {
		outc = nf - 1 - c;
	} else {
		outc = c;
	}


	// Easy way of expanding the time flagging by 1. Useful for killing dropouts. ACES-209
	bool last_sample_ok = true;
	float block_dm0thresh = dm0_thresh/sqrtf((float) nt);
	rescale_dtype dm0min = dm0statarr[ibeam + 1]; // broadcast read. This is to catch dropouts

	for (int t = 0; t < nt; ++t) {
		int inidx = c + nf*(t + nt*ibeam);
		int outidx = t + nt*(outc + nf*ibeam);
		// coalesced read from global
		rescale_dtype vin = (rescale_dtype)inarr[inidx]; // read from global
		rescale_dtype vout = (vin + offset) * scale;
		if (k == 0) { // If we set the timescale to zero, we just don't do any decaying
			decay_offset = 0;
		} else {
			decay_offset = (vout + decay_offset*k)/(1.0 + k);
		}
		rescale_dtype sout = vout - decay_offset;
		int dm0idx = t + nt*ibeam; // DM0 idx: BT order
		rescale_dtype dm0count = dm0countarr[dm0idx];
		rescale_dtype dm0sum = dm0arr[dm0idx] ; // sum accros dm0 - not normalised
		rescale_dtype dm0z = dm0sum*rsqrtf(dm0count);
		rescale_dtype dm0mean = dm0sum/dm0count;
		//int this_sample_ok = fabs(dm0) < dm0_thresh && fabs(sout) < cell_thresh && fabs(dm0sum) < block_dm0thresh;
		bool this_sample_ok = fabs(dm0z) < dm0_thresh && fabs(sout) < cell_thresh && dm0min > -3*dm0_thresh;
		//int this_sample_ok = fabs(dm0) < dm0_thresh && fabs(sout) < cell_thresh;
		if (this_sample_ok && last_sample_ok) {
			sum += vin;
			sum2 += vin*vin;
			sum3 += vin*vin*vin;
			sum4 += vin*vin*vin*vin;
			// non-coalesced write (transpose. Sorry)
			if (subtract_dm0) {
				outarr[outidx] = sout - dm0mean;
			} else {
				outarr[outidx] = sout;
			}
			nsamps += 1;
		} else {
			printf("NOK ibeam/c/t %d/%d/%d dm0/sout/dm0min %f/%f/%f flags %d/%d/%d\n", ibeam, c, t,
					fabs(dm0z), fabs(sout), dm0min,
					fabs(dm0z) < dm0_thresh,
					fabs(sout) < cell_thresh,
					dm0min > -3*dm0_thresh);
			outarr[outidx] = 0.0;
		}

		last_sample_ok = this_sample_ok;

	}

	// write everything back to global memory -- all coalesced
	sumarr[rsidx] = sum;
	sum2arr[rsidx] = sum2;
	sum3arr[rsidx] = sum3;
	sum4arr[rsidx] = sum4;
	decay_offsetarr[rsidx] = decay_offset;
	nsampsarr[rsidx] = (float)nsamps;
}

void rescale_update_and_transpose_float_gpu(rescale_gpu_t& rescale, array4d_t& rescale_buf, const uint8_t* read_buf, bool invert_freq, bool subtract_dm0)
{
	int nbeams = rescale_buf.nw;
	int nf = rescale_buf.nx;
	int nt = rescale_buf.nz;

	// Calculate dm0 for flagging
	rescale_calc_dm0_kernel<<<nbeams, 256>>>(
			read_buf,
			rescale.offset.d_device,
			rescale.scale.d_device,
			rescale.dm0.d_device,
			rescale.dm0count.d_device,
			nf, nt,
			rescale.cell_thresh);

	gpuErrchk(hipDeviceSynchronize());

	// Take the mean all the dm0 times into one big number per beam - this is the how we flag
	// short dropouts see ACES-209
	// probably could do this in rescale_calc_dm0_kernel after yu've done it
	// But i Haven't got htere yet.
	rescale_calc_dm0stats_kernel<<<1, nbeams>>>(
			rescale.dm0.d_device,
			rescale.dm0count.d_device,
			rescale.dm0stats.d_device,
			nt);

	gpuErrchk(hipDeviceSynchronize());

	rescale_update_and_transpose_float_kernel<<<nbeams, nf>>>(
			read_buf,
			rescale.sum.d_device,
			rescale.sum2.d_device,
			rescale.sum3.d_device,
			rescale.sum4.d_device,
			rescale.decay_offset.d_device,
			rescale.nsamps.d_device,
			rescale.offset.d_device,
			rescale.scale.d_device,
			rescale.dm0.d_device,
			rescale.dm0count.d_device,
			rescale.dm0stats.d_device,
			rescale_buf.d_device,
			rescale.decay_constant,
			rescale.dm0_thresh,
			rescale.cell_thresh*rescale.target_stdev,
			nt,
			invert_freq,
			subtract_dm0);
	rescale.sampnum += nt;
	gpuErrchk(hipDeviceSynchronize());

}

__global__ void rescale_update_scaleoffset_kernel (
		rescale_dtype* __restrict__ sum,
		rescale_dtype* __restrict__ sum2,
		rescale_dtype* __restrict__ sum3,
		rescale_dtype* __restrict__ sum4,
		rescale_dtype* __restrict__ meanarr,
		rescale_dtype* __restrict__ stdarr,
		rescale_dtype* __restrict__ kurtarr,
		rescale_dtype* __restrict__ offsetarr,
		rescale_dtype* __restrict__ scalearr,
		rescale_dtype* nsamparr,
		rescale_dtype target_stdev,
		rescale_dtype target_mean,
		rescale_dtype mean_thresh,
		rescale_dtype std_thresh,
		rescale_dtype kurt_thresh,
		int flag_grow)
{
	int c = threadIdx.x;
	int nf = blockDim.x;
	int ibeam = blockIdx.x;
	int i = c + nf*ibeam;
	rescale_dtype nsamp = nsamparr[i];
	rescale_dtype mean = sum[i]/nsamp;
	rescale_dtype mean2 = sum2[i]/nsamp;
	rescale_dtype mean3 = sum3[i]/nsamp;
	rescale_dtype mean4 = sum4[i]/nsamp;
	rescale_dtype variance = mean2 - mean*mean;

	// Excess Kurtosis is k = E([X-mu]**4)/(Var[X]**2) - 3
	// numerator = E[X**4] - 4E[X][E[X**3] + 6 E[X**2]E[X]**2 - 3E[X]**4
	rescale_dtype kurt = (mean4 - 4*mean*mean3 + 6*mean2*mean*mean - 3*mean*mean*mean*mean)/(variance*variance) -3 ;
	// save flag inputs
	meanarr[i] = mean;
	stdarr[i] = sqrtf(variance);
	kurtarr[i] = kurt;

	__syncthreads();
	rescale_dtype scale = 0.0, offset = 0.0;
	int icstart = max(0, c - flag_grow) + nf*ibeam;
	int icend = min(nf, c + flag_grow) + nf*ibeam;
	int flag = 0;
	rescale_dtype expected_mean = 128.;
	rescale_dtype expected_std = 18.;
	for (int ic = icstart; ic < icend; ++ic) {
		rescale_dtype meanoff = fabs(meanarr[ic] - expected_mean);
		rescale_dtype stdoff = fabs(stdarr[ic] - expected_std);
		rescale_dtype kurtoff = fabs(kurtarr[ic]);

		if (nsamp > 0 && (meanoff > mean_thresh ||
				stdoff > std_thresh ||
				kurtoff > kurt_thresh)) {
			flag = 1;
			break;
		}
	}

	if (flag) {
		scale = 0.0;
		offset = 0.0;
	} else {
		if (nsamp == 0) { // Don't update the scale and offset if everything has been flagged
			offset = offsetarr[i];
			scale = scalearr[i];
		} else if (variance == 0.0) {
			scale = 1.0;
			offset = -mean + target_mean/scale;
		} else {
			scale = target_stdev / sqrt(variance);
			offset = -mean + target_mean/scale;
		}
	}

	offsetarr[i] = offset;
	scalearr[i] = scale;

	// reset values to zero
	sum[i] = 0.0;
	sum2[i] = 0.0;
	sum3[i] = 0.0;
	sum4[i] = 0.0;
	nsamparr[i] = 0;
}
void rescale_update_scaleoffset_gpu(rescale_gpu_t& rescale)
{
	assert(rescale.interval_samps > 0);
	int nthreads = rescale.nf;
	assert(rescale.num_elements % nthreads == 0);
	int nblocks = rescale.num_elements / nthreads;
	rescale_update_scaleoffset_kernel<<<nblocks, nthreads>>>(
			rescale.sum.d_device,
			rescale.sum2.d_device,
			rescale.sum3.d_device,
			rescale.sum4.d_device,
			rescale.mean.d_device,
			rescale.std.d_device,
			rescale.kurt.d_device,
			rescale.offset.d_device,
			rescale.scale.d_device,
			rescale.nsamps.d_device,
			rescale.target_stdev,
			rescale.target_mean,
			rescale.mean_thresh,
			rescale.std_thresh,
			rescale.kurt_thresh,
			rescale.flag_grow);
	gpuErrchk(hipDeviceSynchronize());
	rescale.sampnum = 0;
}

