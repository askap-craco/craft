#include "hip/hip_runtime.h"
/*
 * CandidateList.cpp
 *
 *  Created on: 10 Feb 2017
 *      Author: ban115
 */

#include "CandidateList.h"
#include "cuda_utils.h"
#include <stdlib.h>

__global__ void printit(unsigned int *t) {
	printf("It was %d\n", *t);
}

CandidateList::CandidateList(unsigned int max_cand) {
	// Works using unified memory
	gpuErrchk(hipHostMalloc(&m_ncand, sizeof(unsigned int)));
	gpuErrchk(hipHostMalloc(&m_max_cand, sizeof(unsigned int)));
	gpuErrchk(hipHostMalloc(&m_candidates, max_cand * sizeof(candidate_t)));
	*m_max_cand = max_cand;
	clear();
}

//CandidateList::CandidateList(const& CandidateList other) :
//		m_max_cand(other.m_max_cand),
//		m_ncand(other.m_ncand),
//		m_candidates(other.m_candidates)
//{
//
//}


CandidateList::~CandidateList() {
	gpuErrchk(hipHostFree(m_candidates));
	gpuErrchk(hipHostFree(m_ncand));
	gpuErrchk(hipHostFree(m_max_cand));
}

void __host__ CandidateList::clear() {
	// gpuErrchk(hipMemset(m_ncand, 0, sizeof(int)));
	*m_ncand = 0;
}

unsigned int __host__ CandidateList::ncand() {
	return *m_ncand;
}

__host__ unsigned int CandidateList::copy_to_sink(CandidateSink& sink, size_t sampno) {
	unsigned int ncand = CandidateList::ncand();
	for (unsigned int i = 0; i < ncand; ++i) {
		candidate_t* c = &m_candidates[i];
		sink.add_candidate(c->ibeam, c->idt, sampno+ c->t, c->ibc, c->sn);
	}

	return ncand;
}

__device__ unsigned int CandidateList::add_candidate(int ibeam, int idt, int t, int ibc,
		float sn) {
	candidate_t* c = m_candidates + *m_ncand;
	c->ibeam = ibeam;
	c->idt = idt;
	c->t = t;
	c->ibc = ibc;
	c->sn = sn;
	return atomicInc(m_ncand, *m_max_cand);
}
