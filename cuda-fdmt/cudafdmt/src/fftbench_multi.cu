/*
 * fftbench.cu
 *
 *  Created on: 17 Sep 2018
 *      Author: ban115

 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "cuda_utils.h"
#include "cufft_utils.h"


//typedef half2 intype;
//typedef half outtype;

//typedef hipfftComplex intype;
//typedef hipfftReal outtype;
//typedef hipfftComplex ftype;

template <class intype>
void timefft(int n, int batch, hipDataType itype, hipDataType etype, hipDataType otype, bool inplace)
{
  CudaTimer tgpu;
  CpuTimer t;
  
	//intype *data, *out_data;
	hipfftHandle plan;
	size_t data_size=sizeof(intype)*n*(n/2 + 1)*batch;
	int ngpus = 4;
	int gpus[]  = { 0,1,2,3 };
	/*
	for (int i = 0; i < ngpus; i++) {
	  gpuErrchk(hipSetDevice(gpus[i]))
	  gpuErrchk(hipMalloc((void**) &data, data_size));
	  if (inplace) {
	    out_data = data;
	  } else {
	    gpuErrchk(hipMalloc((void**) &out_data, data_size));
	  }
	}
	*/

	hipfftXtSubFormat format;
	if (inplace) {
	  format = HIPFFT_XT_FORMAT_INPLACE;
	} else {
	  format = HIPFFT_XT_FORMAT_INPUT; // THere is also _OUTPU
	}

	long long int nsize[] = {n,n };

		/*cufftSafeCall(hipfftPlanMany(&plan, 2, n,
			NULL, 1, 0, // Simple input layout
			NULL, 1, 0, // Simple output layout
			HIPFFT_C2R, BATCH));
	*/
	size_t worksize[4];
	hipLibXtDesc* in_data;
	hipLibXtDesc* out_data;
	cufftSafeCall(hipfftCreate(&plan));
	if (ngpus > 1) {
	  cufftSafeCall(hipfftXtSetGPUs(plan, ngpus, gpus));
	}
	///cufftSafeCall(hipfftSetAutoAllocation());

	//hipDataType itype = HIP_C_16F;
	//hipDataType etype = HIP_C_16F;
	//hipDataType otype = HIP_R_16F;
	cufftSafeCall(hipfftXtMakePlanMany(plan, 2, nsize,
					  NULL, 1, 256, itype,
					  NULL, 1, 256, otype,
					  batch, worksize, etype
					  ));

	cufftSafeCall(hipfftXtMalloc(plan, (hipLibXtDesc **)&in_data, format));
	if (inplace) {
	  out_data = in_data;
	} else {
	  cufftSafeCall(hipfftXtMalloc(plan, (hipLibXtDesc **)&out_data, format));
	}
	
	// warm up
	//cufftSafeCall(hipfftXtExec(plan, in_data, out_data, HIPFFT_BACKWARD));
	cufftSafeCall(hipfftXtExecDescriptorC2R(plan, in_data,  out_data));

	int niter = 100;
	t.start();
	tgpu.start();
	for (int i = 0; i < niter; ++i) {
	  cufftSafeCall(hipfftXtExecDescriptorC2R(plan, in_data,  out_data));
		//cufftSafeCall(hipfftExecC2R(plan, data, (outtype*) data));
		//cufftSafeCall(hipfftXtExec(plan, in_data, out_data, HIPFFT_BACKWARD));
	}
	tgpu.stop();
	t.stop();
	cout << tgpu << endl;
	//cout << "CPU ONLY" << t << endl;

	float tavg_us = tgpu.get_average_time() / float(niter*batch) * 1e3f;

	printf("%dx%d FFT batch=%d data=%d MB in-place=%d type=%d-> %d. Worksize=%d MB: %f microseconds/FFT= %f k FFTs/sec\n",
	       n,n,batch,data_size/1024/1024, inplace, itype,otype, worksize[0]/1024/1024, tavg_us, 1./tavg_us*1e6f/1e3f);

	cufftSafeCall(hipfftXtFree(in_data));
	if (! inplace) {
	  cufftSafeCall(hipfftXtFree(out_data));
	}

	cufftSafeCall(hipfftDestroy(plan));
}

int main(int argc, char* argv[])
{

	if (argc != 5) {
		printf("%s Usage: gpuid N batchmin batchmax\n", argv[0]);
		return EXIT_FAILURE;
	}

	int cuda_device = atoi(argv[1]);
	hipDeviceProp_t p;
	gpuErrchk(hipGetDeviceProperties(&p, cuda_device));

	printf("FFT Benchmark \n");
	printf("Device[%d]=%s v%d.%d Mem=%d GB shmem/block=%d constmem=%d Warp=%d Clock=%d MHz %d multiprocessors\n",
			cuda_device, p.name, p.major, p.minor, p.totalGlobalMem/1024/1024/1024, p.sharedMemPerBlock,
			p.totalConstMem, p.warpSize, p.clockRate/1000, p.multiProcessorCount
			);

	gpuErrchk( hipSetDevice(cuda_device));

	int n = atoi(argv[2]);
	int batchmin = atoi(argv[3]);
	int batchmax = atoi(argv[4]);
	hipDataType itype = HIP_C_32F;
	hipDataType etype = HIP_C_32F;
	hipDataType otype = HIP_R_32F;

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<hipfftComplex>(n,batch,itype, etype, otype, false);
	}

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<hipfftComplex>(n,batch,itype, etype, otype, true);
	}


	itype = HIP_C_16F;
	etype = HIP_C_16F;
	otype = HIP_R_16F;

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<half2>(n,batch,itype, etype, otype, false);
	}

	for (int batch2 = batchmin; batch2 < batchmax; batch2++) {
		int batch = 1 << batch2;
		timefft<half2>(n,batch,itype, etype, otype, true);
	}


	printf("Benchmark finished\n");
}



