/*
 * craftbench.cu
 *
 *  Created on: 17 Sep 2018
 *      Author: ban115
 *  Updated on August 2019 by DEN15C 
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>  
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include "CudaTimer.h"
#include "CpuTimer.h"
#include "cuda_utils.h"
#include "cufft_utils.h"


void usage()
{
  fprintf(stdout,
	  "craftbench - To check the profermance of CRAFT essentials with different configurations\n"
	  "\n"
	  "Usage: craftbench [options]\n"
	  " -a The ID of GPU to be used \n"
	  " -b FFT size\n"
	  " -c Min number of batch to do, power of 2\n "
	  " -d Max number of batch to do, power of 2\n"
	  " -e Options we want to do with: 0 FFT without host, 1 FFT with memcpy between device and host, more options to be added\n"
	  );
}

template <class intype>
int timefft(int n, int batch, bool inplace, int option)
{
  CudaTimer t;
  intype *data, *out_data, *data_host, *out_data_host;
  hipfftHandle plan;
  size_t data_size=sizeof(intype)*n*(n/2 + 1)*batch;
  hipDataType itype, etype, otype;
  
  if(sizeof(intype) == sizeof(hipfftComplex))
    {
      itype = HIP_C_32F;
      etype = HIP_C_32F;
      otype = HIP_R_32F;
    }
  if(sizeof(intype) == sizeof(half2))
    {
      itype = HIP_C_16F;
      etype = HIP_C_16F;
      otype = HIP_R_16F;
    }
 
  /* Get memory on device */
  gpuErrchk(hipMalloc((void**) &data, data_size));
  if (inplace)
    out_data = data;
  else
    gpuErrchk(hipMalloc((void**) &out_data, data_size));

  /* Get memory on host */
  if(option != 0)
    {
      gpuErrchk(hipHostMalloc((void**) &data_host, data_size));
      if (inplace)
	out_data_host = data_host;
      else
	gpuErrchk(hipHostMalloc((void**) &out_data_host, data_size));
    }
  long long int nsize[] = {n,n };

  /* Create the FFT plan for multiple data types */
  size_t worksize;
  cufftSafeCall(hipfftCreate(&plan));
  cufftSafeCall(hipfftXtMakePlanMany(plan, 2, nsize,
				    NULL, 1, 0, itype,
				    NULL, 1, 0, otype,
				    batch, &worksize, etype
				    ));
  
  /* Warm up and do the real thing */
  cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));
  int i, niter = 1000;
  for (i = 0; i < niter; ++i)
    {
      t.start();
      //cufftSafeCall(hipfftExecC2R(plan, data, (outtype*) data));
      if(option == 1)
	gpuErrchk(hipMemcpy(data, data_host, data_size, hipMemcpyHostToDevice));
      cufftSafeCall(hipfftXtExec(plan, data, out_data, HIPFFT_BACKWARD));
      if(option == 1)
	gpuErrchk(hipMemcpy(out_data_host, out_data, data_size, hipMemcpyDeviceToHost));
      t.stop(); 
    }

  /* Check the timer and display */
  float tavg_us = t.get_average_time() / float(batch) * 1e3f;
  fprintf(stdout, "%dx%d FFT batch=%d data=%d MB in-place=%d type=%d-> %d. Worksize=%d MB: %f microseconds/FFT= %f k FFTs/sec total=%0.2fs\n",
	  n,n,batch,data_size/1024/1024, inplace, itype,otype, worksize/1024/1024, tavg_us, 1./tavg_us*1e6f/1e3f);

  /* Destroy FFT plan */
  cufftSafeCall(hipfftDestroy(plan));

  /* Free device memory */
  gpuErrchk(hipFree(data));
  if (! inplace)
    gpuErrchk(hipFree(out_data));

  /* Free host memory */
  if(option != 0)
    {
      gpuErrchk(hipHostFree(data_host));
      if (! inplace)
	gpuErrchk(hipHostFree(out_data_host));
    }
  
  return EXIT_SUCCESS;
}

// ./craftbench -a 0 -b 256 -c 10 -d 12 -e 0
int main(int argc, char* argv[])
{
  int arg;
  int cuda_device, n, batchmax, batchmin, option;
  int narg = 0, narg_expect = 5;
  
  /* read in argument from command line */
  while((arg=getopt(argc,argv,"a:b:c:hd:e:")) != -1)
    {
      switch(arg)
	{
	case 'h':
	  usage();
	  exit(EXIT_FAILURE);
	  
	case 'a':	  	  
	  if(sscanf(optarg, "%d", &cuda_device) != 1)
	    {
	      usage();	      
	      exit(EXIT_FAILURE);
	    }
	  narg++;
	  break;
	  
	case 'b':	  	  
	  if(sscanf(optarg, "%d", &n) != 1)
	    {
	      usage();	      
	      exit(EXIT_FAILURE);
	    }
	  narg++;
	  break;
	  
	case 'c':	  	  
	  if(sscanf(optarg, "%d", &batchmin) != 1)
	    {
	      usage();	      
	      exit(EXIT_FAILURE);
	    }
	  narg++;
	  break;
	  
	case 'd':	  	  
	  if(sscanf(optarg, "%d", &batchmax) != 1)
	    {
	      usage();	      
	      exit(EXIT_FAILURE);
	    }
	  narg++;
	  break;
	  
	case 'e':	  	  
	  if(sscanf(optarg, "%d", &option) != 1)
	    {
	      usage();	      
	      exit(EXIT_FAILURE);
	    }
	  narg++;
	  break;	  
	}
    }

  /* To check if we et all information */
  if (narg != narg_expect)
    {
      usage();	      
      exit(EXIT_FAILURE);
    }

  /* Check available GPU and give information */
  hipDeviceProp_t p;
  gpuErrchk(hipGetDeviceProperties(&p, cuda_device));  
  fprintf(stdout, "FFT Benchmark \n");
  fprintf(stdout, "Device[%d]=%s v%d.%d Mem=%d GB shmem/block=%d constmem=%d Warp=%d Clock=%d MHz %d multiprocessors\n",
	 cuda_device, p.name, p.major, p.minor, p.totalGlobalMem/1024/1024/1024, p.sharedMemPerBlock,
	 p.totalConstMem, p.warpSize, p.clockRate/1000, p.multiProcessorCount
	 );  
  gpuErrchk(hipSetDevice(cuda_device));

  /* Do the real work */
  int batch2, batch;
  for (batch2 = batchmin; batch2 < batchmax; batch2++)
    {
      batch = 1 << batch2;
      timefft<hipfftComplex>(n,batch,false, option);
    }
  
  for (batch2 = batchmin; batch2 < batchmax; batch2++)
    {
      batch = 1 << batch2;
      timefft<hipfftComplex>(n,batch,true, option);
    }
    
  for (batch2 = batchmin; batch2 < batchmax; batch2++)
    {
      batch = 1 << batch2;
      timefft<half2>(n,batch,false, option);
    }
  
  for (batch2 = batchmin; batch2 < batchmax; batch2++)
    {
      batch = 1 << batch2;
      timefft<half2>(n,batch,true, option);
    }
  
  fprintf(stdout, "Benchmark finished\n");

  return EXIT_SUCCESS;
}
