/*
 * array.c
 *
 *  Created on: 4 Oct 2016
 *      Author: ban115
 */

#include "array.h"
#include <stdarg.h>


int arraynd_idx(const arraynd_t *a, ...)
{
	va_list argp;
	va_start(argp, a);
	assert(a->ndim >= 1);
	assert(a->ndim <= MAX_DIMS);
	int idx = 0;
	for (int d = 1; d <= a->ndim; d++) {
		int i = va_arg(argp, int);
		int dimsize = a->shape[d];
		assert(i >= 0);
		assert(i < dimsize);
		if (d < a->ndim) {
			int next_dim_size = a->shape[d+1];
			idx = i + next_dim_size*idx;
		} else {
			idx += i;
		}
	}

	va_end(argp);
	return idx;
}

__host__ __device__ int array4d_idx(const array4d_t* a, int w, int x, int y, int z)
{
  //assert(w >=0 && w < a->nw);
//  assert(x >=0 && x < a->nx);
//  assert(y >=0 && y < a->ny);
//  assert(z >=0 && z < a->nz);
  int idx = z + a->nz*(y + a->ny*(x + w*a->nx));
  return idx;
}

__host__ __device__ int array4d_idx(int nw, int nx, int ny, int nz, int w, int x, int y, int z)
{
	  int idx = z + nz*(y + ny*(x + w*nx));
	  return idx;
}

__host__ __device__ size_t array4d_size(const array4d_t* a)
{
	return a->nw * a->nx * a->ny * a->nz;
}

__host__ __device__ size_t array2d_size(const array2d_t* a)
{
	return  a->nx * a->ny;
}

size_t array4d_malloc_hostonly(array4d_t* a)
{
	size_t size = array4d_size(a);
	a->d = (fdmt_dtype*) malloc(size*sizeof(fdmt_dtype));
	assert(a->d != NULL);
	return size;
}

size_t array4d_malloc(array4d_t* a, bool host, bool device)
{
	size_t size = 0;
	if (host) {
		size = array4d_malloc_hostonly(a);
	} else {
		a->d = NULL;
	}

	if (device) {
		size  = array4d_size(a);
		size_t free, total;
		gpuErrchk(hipMemGetInfo(&free, &total));

		printf("Allocating [%d, %d, %d, %d] %d MIB total %d/%d on GPU\n",a->nw, a->nx, a->ny, a->nz,
				size*sizeof(fdmt_dtype)/1024/1024, (total-free)/1024/1024, total/1024/1024);

		gpuErrchk( hipMalloc((void**) &a->d_device,
				size*sizeof(fdmt_dtype) ));
		gpuErrchk(hipMemGetInfo(&free, &total));

		printf("Allocated [%d, %d, %d, %d] %d MIB total %d/%d on GPU\n",a->nw, a->nx, a->ny, a->nz,
		size*sizeof(fdmt_dtype)/1024/1024, (total-free)/1024/1024, total/1024/1024);
	} else {
		a->d_device = NULL;
	}
    return size;
}



int array2d_malloc_hostonly(array2d_t* a)
{
	int size = array2d_size(a);
	a->d = (fdmt_dtype*) malloc(size*sizeof(fdmt_dtype));
    assert(a->d != NULL);
    gpuErrchk( hipMalloc((void**) &a->d_device, size*sizeof(fdmt_dtype) ));
    return size;
}

int array2d_malloc(array2d_t* a)
{
	int size = array2d_size(a);
	a->d = (fdmt_dtype*) malloc(size*sizeof(fdmt_dtype));
    assert(a->d != NULL);
    gpuErrchk( hipMalloc((void**) &a->d_device, size*sizeof(fdmt_dtype) ));
    return size;
}

int array4d_copy_to_host(array4d_t* a)
{
	size_t size = array4d_size(a);
	assert(a->d != NULL);
	assert(a->d_device != NULL);
	gpuErrchk(hipMemcpy(a->d, a->d_device, size*sizeof(fdmt_dtype), hipMemcpyDeviceToHost));
	return size;
}

int array4d_cuda_memset(array4d_t*a, char c) {
	size_t size = array4d_size(a);
	gpuErrchk(hipMemset(a->d_device, c, size*sizeof(fdmt_dtype)));
	return size;
}

int array2d_copy_to_device(array2d_t* a)
{
	size_t size = array2d_size(a);
	gpuErrchk(hipMemcpy(a->d_device, a->d, size*sizeof(fdmt_dtype), hipMemcpyHostToDevice));
	return size;
}

int array4d_copy_to_device(array4d_t* a)
{
	size_t size = array4d_size(a);
	assert(a->d_device != NULL);
	assert(a->d != NULL);
	gpuErrchk(hipMemcpy(a->d_device, a->d, size*sizeof(fdmt_dtype), hipMemcpyHostToDevice));
	return size;
}

int array3d_idx(const array3d_t* a, int x, int y, int z)
{
  assert(x >=0 && x < a->nx);
  assert(y >=0 && y < a->ny);
  assert(z >=0 && z < a->nz);
  int idx = z + a->nz*(y + a->ny*x);
  return idx;
}

int array2d_idx(const array2d_t* a, int x, int y)
{
  assert(x >=0 && x < a->nx);
  assert(y >=0 && y < a->ny);
  if (!(y >=0 && y < a->ny)) {
  }

  int idx = y + a->ny*x;

  return idx;
}

int array2d_dump(const array2d_t* a, const char* foutname)
{
  FILE* fout = fopen(foutname, "w");
  fwrite(&a->nx, sizeof(int), 1, fout);
  fwrite(&a->ny, sizeof(int), 1, fout);
  fwrite(a->d, sizeof(fdmt_dtype), a->nx*a->ny, fout);
  fclose(fout);

  return 0;
}

int array3d_dump(const array3d_t* a, const char* foutname)
{
  FILE* fout = fopen(foutname, "w");
  fwrite(&a->nx, sizeof(int), 1, fout);
  fwrite(&a->ny, sizeof(int), 1, fout);
  fwrite(&a->nz, sizeof(int), 1, fout);
  fwrite(a->d, sizeof(fdmt_dtype), a->nx*a->ny*a->nz, fout);
  fclose(fout);
  return 0;
}

void array4d_print_shape(const array4d_t* a)
{
	printf("nw=%d nx=%d ny=%d nz=%d\b", a->nw, a->nx, a->ny, a->nz);
}

void array4d_set(array4d_t* a, fdmt_dtype v)
{
	assert(a->d != NULL);
	for(int i = 0; i < array4d_size(a); ++i) {
		a->d[i] = v;
	}
	array4d_copy_to_device(a);
}

size_t array4d_zero(array4d_t* a) {
	size_t size = array4d_size(a);
	if (a->d_device) {
		gpuErrchk(hipMemset(a->d_device, size*sizeof(fdmt_dtype), 0));
	}
	if (a->d) {
		bzero(a->d, size*sizeof(fdmt_dtype));
	}

	return size;
}

int array4d_dump(const array4d_t* a, const char* foutname)
{
  FILE* fout = fopen(foutname, "w");
  fwrite(&a->nw, sizeof(int), 1, fout);
  fwrite(&a->nx, sizeof(int), 1, fout);
  fwrite(&a->ny, sizeof(int), 1, fout);
  fwrite(&a->nz, sizeof(int), 1, fout);
  fwrite(a->d, sizeof(fdmt_dtype), a->nw*a->nx*a->ny*a->nz, fout);
  fclose(fout);
  return 0;
}
